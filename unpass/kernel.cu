#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <stdio.h>

#define PASSWORD_LEN		(14)
#define PASSWORD_LEN_MAX	(16)

// �p�X���[�h�̌v�Z�J�[�l��
__global__ void checkPassKernel(
	unsigned char *result,			// OUTPUT	�p�X������TRUE��Ԃ���	[x*y]
	const unsigned char*candidate,	// INPUT	�p�X���[�h���(�����R�[�h�ϊ��ς�) [x*y][16]
	const unsigned char*refer_chk	// INPUT	����p�`�F�b�N�f�B�W�b�g	[x*y][16]
)
{
	bool result_b(false), match_flg;
	int a(0), c(0), cb(0), chridx, strlength;
	unsigned char mem20(0), mem21(0);
	unsigned char mem50(0), mem51(0), mem52(0), mem53(1), mem54(0);

	int th_x = threadIdx.x;		// 
//	int th_y = threadIdx.y;

	auto ror = [&c](unsigned char v)
	{
		unsigned char wc = c;

		c = v & 0x01;
		return (unsigned char)((v >> 1) | (wc << 7));
	};
	auto adc = [&c](unsigned char lhs, unsigned char rhs)
	{
		unsigned short v;

		v = (unsigned short)lhs + (unsigned short)rhs + (unsigned short)c;
		c = (v > 0xFF) ? 1 : 0;

		return (v & 0xFF);
	};
	auto bitcnt = [](unsigned long int v)
	{
			v = (v & 0x55555555) + (v >> 1 & 0x55555555);
			v = (v & 0x33333333) + (v >> 2 & 0x33333333);
			v = (v & 0x0f0f0f0f) + (v >> 4 & 0x0f0f0f0f);
			v = (v & 0x00ff00ff) + (v >> 8 & 0x00ff00ff);
		return  (v & 0x0000ffff) + (v >> 16 & 0x0000ffff);
	};

	strlength = refer_chk[2];
	for (chridx = 0; chridx < strlength; ++chridx) {
		a = candidate[(th_x * PASSWORD_LEN_MAX) + chridx];

		// calc checkdigit1	(���[�v�W�J���Ă���)
//		for (int bit = 7; bit >= 0; --bit) {
//			c = (a >> bit) & 1;
//			mem20 = ror(mem20);
//			mem21 = ror(mem21);
//
//			if (c) {
//				mem20 ^= 0x84;
//				mem21 ^= 0x08;
//			}
//		}
		c = ((a & 0x80) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x40) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x20) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x10) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x08) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x04) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x02) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x01) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		// calc checkdigit2
		c = (mem20 >= 0xE5) ? 1 : 0;
		mem50 = adc(a, mem50);
		mem51 = adc(mem51, mem21);
		// calc checkdigit3
		mem52 ^= a;
		// calc checkdigit4
		{
			unsigned char v = ror(mem53);
			mem53 = adc(v, a);
		}
		// calc checkdigit5
		mem54 += (unsigned char)(c + bitcnt(a));
	}
	
	result_b = true;
	match_flg = (mem20 == refer_chk[0]);
	result_b = result_b && match_flg;
	match_flg = (mem21 == refer_chk[1]);
	result_b = result_b && match_flg;
	match_flg = (mem50 == refer_chk[3]);
	result_b = result_b && match_flg;
	match_flg = (mem51 == refer_chk[4]);
	result_b = result_b && match_flg;
	match_flg = (mem52 == refer_chk[5]);
	result_b = result_b && match_flg;
	match_flg = (mem53 == refer_chk[6]);
	result_b = result_b && match_flg;
	match_flg = (mem54 == refer_chk[7]);
	result_b = result_b && match_flg;

	result[th_x] = result_b;
}


// 2�����̃p�X���[�h���𐶐�����
__global__ void chrconvKernel(
	unsigned char *candidate,			// OUTPUT	����������f�[�^
	const unsigned char *basecode,		// INPUT	�Œ蕔���Ƃ��ĎQ�Ƃ��錅���[16]
	const unsigned char *tbl_chrdict,	// INPUT	�����R�[�h�e�[�u��[256]
	const unsigned int	passlen,		// INPUT	�ϊ��Ώۂ̌���(�S��)
	const unsigned int	dictlen			// INPUT	�����퐔
)
{
	int th_x = threadIdx.x;

	for (int i = 0; i < (passlen - 1); ++i) {
		candidate[(PASSWORD_LEN_MAX * th_x) + i] = tbl_chrdict[basecode[i]];
	}
}


using namespace std;

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t chkPass(unsigned char *, string, int, unsigned char *);

const unsigned char g_chrcode[256] =
{
	0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0x2D,0,0,0,0,0,0,0,0,0,0,0,0x2B,0,0,
0x25,0x04,0x0C,0x14,0x1C,0x24,0x05,0x0D,0x15,0x1D,0,0,0,0,0,0,
0,0x00,0x08,0x10,0x18,0x20,0x28,0x30,0x01,0x09,0x11,0x19,0x21,0x29,0x31,0x02,
0x0A,0x12,0x1A,0x22,0x2A,0x32,0x03,0x0B,0x13,0x1B,0x23,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0x33,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0x35,0,0,0,0,0,
0,0,0,0,0,0x2C,0,0,0,0,0,0,0,0,0,0,
0,0x34,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
};

//const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ" };
const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!.-nmc�Ѻ" };


int main()
{
	unsigned char	pass_columns[PASSWORD_LEN_MAX] = { 0 };
	int passed = -1;

	hipError_t cudaStatus;

	const size_t	dictlen = sizeof(_dict);
	std::string chrtable(_dict, dictlen);
	unsigned char *chkresult = new unsigned char[dictlen * dictlen];

	const int		COLUMNS = 14;
	const int		refresh_col_n = 4;
	const int		chkcol = 8;
	unsigned char	chkkey;
	unsigned char	refresh_col;		// �i���\���p�̔��茅

	// �I������p���l��ޔ�
	chkkey = pass_columns[chkcol];
	refresh_col = pass_columns[refresh_col_n];

	while (pass_columns[chkcol] == chkkey) {

		if (refresh_col != pass_columns[refresh_col_n]) {

			for (int idx = 0; idx < PASSWORD_LEN; ++idx) {
				fprintf(stderr, "%c", _dict[pass_columns[idx]]);
			}
			fprintf(stderr, "\n");
			refresh_col = pass_columns[refresh_col_n];
		}

		hipError_t cudaStatus = chkPass(chkresult, chrtable, 14, pass_columns);
		if (cudaStatus != hipSuccess) { fprintf(stderr, "passcheck failed!"); return 1; }

		for (int idx = 0; idx < (chrtable.size() * chrtable.size()); ++idx) {
			if (chkresult[idx] != 0) {
				fprintf(stderr, "Passed %d\n", idx);
			}
		}

		// �I�������ɍ��v�����Ԃ𔻒肵����T���I��
		bool carry = true;
		for (int col = (2); col < (COLUMNS - 1); ++col) {
			//	for (int col = 2; col < (COLUMNS - 1); ++col) {
			pass_columns[col] += carry ? 1 : 0;
			carry = (pass_columns[col] == dictlen);
			pass_columns[col] = carry ? 0 : pass_columns[col];
			//	candidate[col] = dict[counters[col]];
		}
	}

	
	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	delete[] chkresult;

	return 0;
}

// �f�o�C�X�Ƃ̂����
hipError_t chkPass(
	unsigned char *result_holder,		// OUTPUT �v�Z���ʂ̎���
	string	chrdic,						// ������̃e�[�u��
	int		passlength,					// �p�X���[�h����
	unsigned char *passcolmns			// �p�X���[�h�̑�������p�J�E���^[PASSWORD_LEN_MAX = 16]
)
{
	hipError_t cudaStatus;

	unsigned char *dev_result = 0;		// ���茋��
	unsigned char *dev_candidate = 0;	// �ƍ��Ώۂ̃L�[�ƒ������̔z��
	unsigned char *dev_reference = 0;	// ����Ώۂ̃`�F�b�N�f�B�W�b�g

	unsigned char *dev_passnum = 0;		// �p�X���[�h�����p�J�E���^
	unsigned char *dev_lut_code = 0;	// �����ϊ��e�[�u��

	int dlen = chrdic.size();

	unsigned char reference[16] = {
		0x65, 0x94, 0x0E, 0xAC, 0xE9, 0x07, 0x33, 0x25,	// ���T���ׂ��`�F�b�N�f�B�W�b�g�Q
		0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00
	};


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	// �ƍ����ʂ͎����̕����핪
	cudaStatus = hipMalloc((void**)&dev_result, chrdic.size() * chrdic.size() * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �p�X���[�h��␔�͕����핪
	cudaStatus = hipMalloc((void**)&dev_candidate, chrdic.size() * chrdic.size() * sizeof(unsigned char) * PASSWORD_LEN_MAX);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �����ϊ��e�[�u��
	cudaStatus = hipMalloc((void**)&dev_lut_code, 256);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �����e�[�u��
	cudaStatus = hipMalloc((void**)&dev_passnum, PASSWORD_LEN_MAX);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �ƍ��Ώۂ̃`�F�b�N�f�B�W�b�g�͈���
	cudaStatus = hipMalloc((void**)&dev_reference, 16);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// Copy input vectors from host memory to GPU buffers.
	// �����R�[�h�ϊ��e�[�u��
	cudaStatus = hipMemcpy(dev_lut_code, g_chrcode, 256, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	// �J�E���^���當���R�[�h����
	chrconvKernel << <chrdic.size(), chrdic.size() >> >	(dev_candidate, dev_passnum, dev_lut_code, passlength, chrdic.size());
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "chrconvKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	cudaStatus = hipMemcpy(dev_reference, reference, 16, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipDeviceSynchronize();

	// Launch a kernel on the GPU with one thread for each element.
	checkPassKernel << <chrdic.size(), chrdic.size() >> > (dev_result, dev_candidate, dev_reference);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result_holder, dev_result, chrdic.size() * chrdic.size(), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

Error:
	hipFree(dev_result);
	hipFree(dev_candidate);
	hipFree(dev_reference);
	hipFree(dev_passnum);
	hipFree(dev_lut_code);

	return(cudaStatus);
}

#if 0
// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<1, size>>>(dev_c, dev_a, dev_b);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
#endif

