#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <stdio.h>

__global__ void calc_1_4col(
	unsigned char *work4col,			// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	const unsigned char *chrmask,		// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const unsigned char *lut_x31F4,		// $31F4���o�p��LUT
	const unsigned char *lut_x31F5		// $31F5���o�p��LUT
)
{
	unsigned int gblidx = 8 * (((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x);

	unsigned char	col1 = blockIdx.y;
	unsigned char	col2 = blockIdx.x >> 6;
	unsigned char	col3 = blockIdx.x & 0x3F;
	unsigned char	col4 = threadIdx.x;

	unsigned char	c = 0;					// �L�����[�t���O

	// �ꕔ�I�y�R�[�h���C�����C����
	auto bitrev = [](unsigned int v) {	unsigned int wk = __brev(v); return (wk >> 24); };
	auto adc = [&c](unsigned short vl, unsigned short vr) { unsigned short wk = vl + vr + c; c = (wk >> 8) & 0x01; return (wk & 0xFF); };
	auto ror = [&c](unsigned char v) { unsigned char wc = c; c = v & 0x01; return (unsigned char)((v >> 1) | (wc << 7)); };
	auto bitcnt = [](unsigned long int v) { return (__popc(v)); };

	unsigned char pre31f4(0), pre31f5(0), pre31f7(0), pre31f8(0), pre31fa(1), pre31fb(0);
	unsigned char xor31f4, xor31f5, wk31fa;

	const unsigned char	unvalid = chrmask[col1] | chrmask[col2] | chrmask[col3] | chrmask[col4];
	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	// col1�`col4�ŗ^�����Ȃ�тŌv�Z���܂킷
	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr1) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr1, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr1);
	pre31fb += c + bitcnt(chr1);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr2) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr2, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr2);
	pre31fb += c + bitcnt(chr2);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr3) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr3, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr3);
	pre31fb += c + bitcnt(chr3);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr4) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr4, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr4);
	pre31fb += c + bitcnt(chr4);

	/*$31F4*/	work4col[gblidx + 0] = pre31f4;
	/*$31F5*/	work4col[gblidx + 1] = pre31f5;
	/*$31F7*/	work4col[gblidx + 2] = pre31f7;
	/*$31F8*/	work4col[gblidx + 3] = pre31f8;
	/*$31F9*/	work4col[gblidx + 4] = unvalid | (chr1 ^ chr2 ^ chr3 ^ chr4);
	/*$31FA*/	work4col[gblidx + 5] = pre31fa;
	/*$31FB*/	work4col[gblidx + 6] = pre31fb;
	/*carry*/	work4col[gblidx + 7] = c;
				// F6�͎����Œ�l�Ȃ̂ł߂Ď��̌��ɓn���L�����[���i�[����
}

// ����܂ł̌��̒l���g���Ēǉ���4�����̌v�Z�l�����߂�
__global__ void calc_4col(
	unsigned char		*work8col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	const unsigned char	*work4col,		// ����܂łɌv�Z���Ă������Z����
	const unsigned char *chrmask,		// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const unsigned char *lut_x31F4,		// $31F4���o�p��LUT
	const unsigned char *lut_x31F5,		// $31F5���o�p��LUT
	const unsigned int	idx4			// work4col�ɑΉ�����C���f�b�N�X�l
)
{
	// �X���b�h�ԍ����珑�����ݐ�C���f�b�N�X���v�Z
	unsigned int gblidx = 8 * (((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x);

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;
	const unsigned int	rdidx = idx4 * 8;

	unsigned char pre31f4(work4col[rdidx + 0]);
	unsigned char pre31f5(work4col[rdidx + 1]);
	unsigned char pre31f7(work4col[rdidx + 2]);
	unsigned char pre31f8(work4col[rdidx + 3]);
	unsigned char pre31f9(work4col[rdidx + 4]);
	unsigned char pre31fa(work4col[rdidx + 5]);
	unsigned char pre31fb(work4col[rdidx + 6]);
	unsigned char	    c(work4col[rdidx + 7]);	// �L�����[�t���O

	unsigned char xor31f4, xor31f5, wk31fa;

	// �ꕔ�I�y�R�[�h���C�����C����
	auto bitrev = [](unsigned int v) {	unsigned int wk = __brev(v); return (wk >> 24); };
	auto adc = [&c](unsigned short vl, unsigned short vr) { unsigned short wk = vl + vr + c; c = (wk >> 8) & 0x01; return (wk & 0xFF); };
	auto ror = [&c](unsigned char v) { unsigned char wc = c; c = v & 0x01; return (unsigned char)((v >> 1) | (wc << 7)); };
	auto bitcnt = [](unsigned long int v) { return (__popc(v)); };

	unsigned char	unvalid = chrmask[col1] | chrmask[col2] | chrmask[col3] | chrmask[col4];
	unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	// col1�`col4�ŗ^�����Ȃ�тŌv�Z���܂킷
	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr1) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr1, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr1);
	pre31fb += c + bitcnt(chr1);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr2) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr2, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr2);
	pre31fb += c + bitcnt(chr2);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr3) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr3, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr3);
	pre31fb += c + bitcnt(chr3);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr4) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr4, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr4);
	pre31fb += c + bitcnt(chr4);


	/*$31F4*/	work8col[gblidx + 0] = pre31f4;
	/*$31F5*/	work8col[gblidx + 1] = pre31f5;
	/*$31F7*/	work8col[gblidx + 2] = pre31f7;
	/*$31F8*/	work8col[gblidx + 3] = pre31f8;
	/*$31F9*/	work8col[gblidx + 4] = unvalid | (pre31f9 ^ chr1 ^ chr2 ^ chr3 ^ chr4);
	/*$31FA*/	work8col[gblidx + 5] = pre31fa;
	/*$31FB*/	work8col[gblidx + 6] = pre31fb;
	/*carry*/	work8col[gblidx + 7] = c;
	// F6�͎����Œ�l�Ȃ̂ł߂Ď��̌��ɓn���L�����[���i�[����
}

// �c��̂P�����o�ƍŏI���܂ł̃`�F�b�N�f�B�W�b�g�v�Z�A�L������
__global__ void calclast_validate(
	unsigned char		*result,	// OUT  14�����̊e�f�[�^�v�Z����
	unsigned char		*valid,		// OUT  �p�X���[�h�Ƃ��Đ��藧�Ȃ�1(true)�łȂ����0(false)
	const unsigned char	*work,		// IN	12���ڂ܂łœ��o���ꂽ�r���l
	const unsigned char *lut_x31F4,	// IN	$31F4���o�p��LUT
	const unsigned char *lut_x31F5,	// IN	$31F5���o�p��LUT
	const unsigned char	*chrmask,	// IN	�L���ȕ����R�[�h���ǂ����̃t���O�}�b�v
	const unsigned int	offset		// IN	blockIdx.x�̃I�t�Z�b�g
)
{
	const unsigned int	gblidx = 8 * blockIdx.x + (offset * gridDim.x);	// �u���b�N�ԍ�x��work�ɑ΂���C���f�b�N�X
	const unsigned char col13 = threadIdx.x;		// �X���b�hidx.x��13���ڂ̕����R�[�h���

	unsigned char pre31f4(work[gblidx + 0]);
	unsigned char pre31f5(work[gblidx + 1]);
	unsigned char pre31f7(work[gblidx + 2]);
	unsigned char pre31f8(work[gblidx + 3]);
	unsigned char pre31f9(work[gblidx + 4]);
	unsigned char pre31fa(work[gblidx + 5]);
	unsigned char pre31fb(work[gblidx + 6]);
	unsigned char	    c(work[gblidx + 7]);	// �L�����[�t���O

	unsigned char xor31f4, xor31f5, wk31fa;

	// �ꕔ�I�y�R�[�h���C�����C����
	auto bitrev = [](unsigned int v) {	unsigned int wk = __brev(v); return (wk >> 24); };
	auto adc = [&c](unsigned short vl, unsigned short vr) { unsigned short wk = vl + vr + c; c = (wk >> 8) & 0x01; return (wk & 0xFF); };
	auto ror = [&c](unsigned char v) { unsigned char wc = c; c = v & 0x01; return (unsigned char)((v >> 1) | (wc << 7)); };
	auto bitcnt = [](unsigned long int v) { return (__popc(v)); };

	unsigned char wk14 = pre31f9 ^ col13;	// 13���ڂ�12���ڂ܂ł̌��ʂ�XOR�� 31f9
	pre31f9 = (pre31f9 & 0x80) | chrmask[wk14 & 0x3F];	// �����R�[�h����͈����p��

	unsigned char	col14 = pre31f9 ^ 0x07;		// ����3bit���]��14����

	const unsigned char chr1(col13 & 0x3F), chr2(col14 & 0x3F);


	// col13�`col14�ŗ^�����Ȃ�тŌv�Z���܂킷
	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr1) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr1, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr1);
	pre31fb += c + bitcnt(chr1);

	xor31f4 = lut_x31F4[pre31f5]; xor31f5 = lut_x31F5[pre31f5];
	pre31f5 = pre31f4 ^ xor31f5;
	pre31f4 = bitrev(chr2) ^ xor31f4;	c = (pre31f4 >= 0xE5) ? 1 : 0;
	pre31f7 = adc(chr2, pre31f7);
	pre31f8 = adc(pre31f8, pre31f5);	wk31fa = ror(pre31fa);
	pre31fa = adc(wk31fa, chr2);
	pre31fb += c + bitcnt(chr2);

	bool	judge = true
		&& (pre31f4 == 0x65)
		&& (pre31f5 == 0x94)
		&& (pre31f7 == 0xAC)
		&& (pre31f8 == 0xE9)
		&& (pre31f9 == 0x07)
		&& (pre31fa == 0x33)
		&& (pre31fb == 0x25)
		;

	/*$31F4*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 0] = pre31f4;
	/*$31F5*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 1] = pre31f5;
	/*$31F7*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 2] = pre31f7;
	/*$31F8*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 3] = pre31f8;
	/*$31F9*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 4] = pre31f9;
	/*$31FA*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 5] = pre31fa;
	/*$31FB*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 6] = pre31fb;
	/*Passed*/	result[((blockIdx.x * blockDim.x) + threadIdx.x * 8) + 7] = judge;

	valid[(blockIdx.x * blockDim.x) + threadIdx.x] = judge;
}

const unsigned char lut_xor31F4[256] = {
	0x00,0x11,0x23,0x32,0x46,0x57,0x65,0x74,0x8C,0x9D,0xAF,0xBE,0xCA,0xDB,0xE9,0xF8,
	0x10,0x01,0x33,0x22,0x56,0x47,0x75,0x64,0x9C,0x8D,0xBF,0xAE,0xDA,0xCB,0xF9,0xE8,
	0x21,0x30,0x02,0x13,0x67,0x76,0x44,0x55,0xAD,0xBC,0x8E,0x9F,0xEB,0xFA,0xC8,0xD9,
	0x31,0x20,0x12,0x03,0x77,0x66,0x54,0x45,0xBD,0xAC,0x9E,0x8F,0xFB,0xEA,0xD8,0xC9,
	0x42,0x53,0x61,0x70,0x04,0x15,0x27,0x36,0xCE,0xDF,0xED,0xFC,0x88,0x99,0xAB,0xBA,
	0x52,0x43,0x71,0x60,0x14,0x05,0x37,0x26,0xDE,0xCF,0xFD,0xEC,0x98,0x89,0xBB,0xAA,
	0x63,0x72,0x40,0x51,0x25,0x34,0x06,0x17,0xEF,0xFE,0xCC,0xDD,0xA9,0xB8,0x8A,0x9B,
	0x73,0x62,0x50,0x41,0x35,0x24,0x16,0x07,0xFF,0xEE,0xDC,0xCD,0xB9,0xA8,0x9A,0x8B,
	0x84,0x95,0xA7,0xB6,0xC2,0xD3,0xE1,0xF0,0x08,0x19,0x2B,0x3A,0x4E,0x5F,0x6D,0x7C,
	0x94,0x85,0xB7,0xA6,0xD2,0xC3,0xF1,0xE0,0x18,0x09,0x3B,0x2A,0x5E,0x4F,0x7D,0x6C,
	0xA5,0xB4,0x86,0x97,0xE3,0xF2,0xC0,0xD1,0x29,0x38,0x0A,0x1B,0x6F,0x7E,0x4C,0x5D,
	0xB5,0xA4,0x96,0x87,0xF3,0xE2,0xD0,0xC1,0x39,0x28,0x1A,0x0B,0x7F,0x6E,0x5C,0x4D,
	0xC6,0xD7,0xE5,0xF4,0x80,0x91,0xA3,0xB2,0x4A,0x5B,0x69,0x78,0x0C,0x1D,0x2F,0x3E,
	0xD6,0xC7,0xF5,0xE4,0x90,0x81,0xB3,0xA2,0x5A,0x4B,0x79,0x68,0x1C,0x0D,0x3F,0x2E,
	0xE7,0xF6,0xC4,0xD5,0xA1,0xB0,0x82,0x93,0x6B,0x7A,0x48,0x59,0x2D,0x3C,0x0E,0x1F,
	0xF7,0xE6,0xD4,0xC5,0xB1,0xA0,0x92,0x83,0x7B,0x6A,0x58,0x49,0x3D,0x2C,0x1E,0x0F,
};

const unsigned char lut_xor31F5[256] = {
	0x00,0x89,0x12,0x9B,0x24,0xAD,0x36,0xBF,0x48,0xC1,0x5A,0xD3,0x6C,0xE5,0x7E,0xF7,
	0x81,0x08,0x93,0x1A,0xA5,0x2C,0xB7,0x3E,0xC9,0x40,0xDB,0x52,0xED,0x64,0xFF,0x76,
	0x02,0x8B,0x10,0x99,0x26,0xAF,0x34,0xBD,0x4A,0xC3,0x58,0xD1,0x6E,0xE7,0x7C,0xF5,
	0x83,0x0A,0x91,0x18,0xA7,0x2E,0xB5,0x3C,0xCB,0x42,0xD9,0x50,0xEF,0x66,0xFD,0x74,
	0x04,0x8D,0x16,0x9F,0x20,0xA9,0x32,0xBB,0x4C,0xC5,0x5E,0xD7,0x68,0xE1,0x7A,0xF3,
	0x85,0x0C,0x97,0x1E,0xA1,0x28,0xB3,0x3A,0xCD,0x44,0xDF,0x56,0xE9,0x60,0xFB,0x72,
	0x06,0x8F,0x14,0x9D,0x22,0xAB,0x30,0xB9,0x4E,0xC7,0x5C,0xD5,0x6A,0xE3,0x78,0xF1,
	0x87,0x0E,0x95,0x1C,0xA3,0x2A,0xB1,0x38,0xCF,0x46,0xDD,0x54,0xEB,0x62,0xF9,0x70,
	0x08,0x81,0x1A,0x93,0x2C,0xA5,0x3E,0xB7,0x40,0xC9,0x52,0xDB,0x64,0xED,0x76,0xFF,
	0x89,0x00,0x9B,0x12,0xAD,0x24,0xBF,0x36,0xC1,0x48,0xD3,0x5A,0xE5,0x6C,0xF7,0x7E,
	0x0A,0x83,0x18,0x91,0x2E,0xA7,0x3C,0xB5,0x42,0xCB,0x50,0xD9,0x66,0xEF,0x74,0xFD,
	0x8B,0x02,0x99,0x10,0xAF,0x26,0xBD,0x34,0xC3,0x4A,0xD1,0x58,0xE7,0x6E,0xF5,0x7C,
	0x0C,0x85,0x1E,0x97,0x28,0xA1,0x3A,0xB3,0x44,0xCD,0x56,0xDF,0x60,0xE9,0x72,0xFB,
	0x8D,0x04,0x9F,0x16,0xA9,0x20,0xBB,0x32,0xC5,0x4C,0xD7,0x5E,0xE1,0x68,0xF3,0x7A,
	0x0E,0x87,0x1C,0x95,0x2A,0xA3,0x38,0xB1,0x46,0xCF,0x54,0xDD,0x62,0xEB,0x70,0xF9,
	0x8F,0x06,0x9D,0x14,0xAB,0x22,0xB9,0x30,0xC7,0x4E,0xD5,0x5C,0xE3,0x6A,0xF1,0x78,
};

using namespace std;

// const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!.-nmc�Ѻ" };
const unsigned char charvalidmask[64] = { 
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,	// 0x06��0x07������
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
};

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
);

const unsigned long	WORKSIZE = 64 * 64 * 64 * 64;
static unsigned char result[WORKSIZE];

int main()
{
	hipError_t cudaStatus;

	chkthread(result, WORKSIZE);

	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	return 0;
}

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
)
{
	hipError_t cudaStatus;
	unsigned char *dev_chrcode_mask = 0;	// �L�������R�[�h����p�}�X�N		�L����0x00�A������0x80�B
	unsigned char *dev_validpass = 0;		// �v�Z�������Ȃ��1������

	unsigned char *dev_31F4_xortbl = 0;		// $31F4���o�p�e�[�u��
	unsigned char *dev_31F5_xortbl = 0;		// $31F5���o�p�e�[�u��

	unsigned char *cpu_validpass = 0;		// �L���p�X���[�h���茋��(CPU��)
	unsigned char *cpu_calcresult = 0;		// �v�Z���ʊm�F�p

	unsigned char *dev_res4 = 0;		// �擪4���Ԃ�̌v�Z���ʂƃL�����[���
	unsigned char *dev_res8 = 0;		// 5-8���܂ł̌v�Z���ʂƃL�����[���
	unsigned char *dev_res12 = 0;		// 9-12���܂Ōv�Z���ʂƃL�����[���
	unsigned char *dev_result = 0;		// 14���̌v�Z����

	unsigned long long validcnt = 0;	// �`�F�b�N�f�B�W�b�g��ʂ����p�X���[�h�̌�

	dim3	 block_1677m(64), grid_1677m(64 * 64, 64);	// ����64�� 4�����̕���v�Z�O���b�h�p	16.8M items

	cudaStatus = hipSetDevice(0);		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	
	// ���o�p�e�[�u��(const)
	cudaStatus = hipMalloc((void**)&dev_chrcode_mask, 64 * sizeof(unsigned char));						if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_31F4_xortbl, 256 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_31F5_xortbl, 256 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 4�����̊e�f�[�^�Z�o�p�̈�m��
	cudaStatus = hipMalloc((void**)&dev_res4, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res8, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res12, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_result, 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_validpass, 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }


	cudaStatus = hipMemcpy(dev_chrcode_mask, charvalidmask, 64 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_31F4_xortbl, lut_xor31F4, 256 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_31F5_xortbl, lut_xor31F5, 256 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_validpass, 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_calcresult, 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �擪�S���S�g�݂��킹�̌v�Z
	calc_1_4col <<< dim3(64 * 64, 64), dim3(64) >>> (dev_res4, dev_chrcode_mask, dev_31F4_xortbl, dev_31F5_xortbl);	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calu14col!\n", cudaStatus); goto Error; }

	validcnt = 0;

	for (int xor4idx = 0; xor4idx < (64 * 64 * 64 * 64); ++xor4idx) {

		// 5-8���̑g�������v�Z����
		calc_4col <<< dim3(64 * 64, 64), dim3(64) >>> (dev_res8, dev_res4, dev_chrcode_mask, dev_31F4_xortbl, dev_31F5_xortbl, xor4idx);
		cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

		for (int xor8idx = 0; xor8idx < (64 * 64 * 64 * 64); ++xor8idx) {
			// 9�`12�����̑g�������v�Z����
			calc_4col <<< dim3(64 * 64, 64), 64 >>> (dev_res12, dev_res8, dev_chrcode_mask, dev_31F4_xortbl, dev_31F5_xortbl, xor8idx);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }


			for (int xor12idx = 0; xor12idx < (64 * 64); ++xor12idx) {
				// 13,14���ڂ̓��o�ƃ`�F�b�N�f�B�W�b�g���ʂ��Ă��邩�̔���
				calclast_validate << < 64 * 64, 64 >> > (dev_result, dev_validpass, dev_res12, dev_31F4_xortbl, dev_31F5_xortbl, dev_chrcode_mask, xor12idx);
				cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching validation!\n", cudaStatus); goto Error; }


				cudaStatus = hipMemcpy(cpu_calcresult, dev_result, size_t(64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				cudaStatus = hipMemcpy(cpu_validpass, dev_validpass, 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
				for (int chkrs = 0; chkrs < (64 * 64 * 64); ++chkrs) {
					printf("\n");

					for (int vidx = 0; vidx < 8; ++vidx) {
						printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
					}
					validcnt += cpu_validpass[chkrs];
				}
//				printf("\n%d items\n", validcnt);


			}
		}
	}
	printf("found \n%lld items\n", validcnt);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(cpu_result, dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }


Error:
	hipFree(dev_chrcode_mask);
	hipFree(dev_validpass);

	hipFree(dev_31F4_xortbl);
	hipFree(dev_31F5_xortbl);

	hipFree(cpu_validpass);
	hipFree(cpu_calcresult);

	hipFree(dev_res4);
	hipFree(dev_res8);
	hipFree(dev_res12);
	hipFree(dev_result);

	return(cudaStatus);

}
