#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <stdio.h>

__device__ inline unsigned char bitrev(unsigned int v)
{
	unsigned int wk = __brev(v);
	return (wk >> 24) & 0xFF;
}

__device__ inline unsigned char adc(unsigned char &c, unsigned char vl, unsigned char vr)
{
	unsigned short wk = vl + vr + c;
	c = (wk & 0x0100) ? 1 : 0;
	return (unsigned char)(wk & 0xFF);
}

__device__ inline unsigned char ror(unsigned char &c, unsigned char v)
{
	unsigned char wc = c * 0x80;
	c = v & 0x01;
	return (unsigned char)((v >> 1) | wc);
}

__device__ inline unsigned char bitcnt(unsigned char v)
{
	return ((unsigned char)__popc(v));
}

__device__ unsigned char calcstep(const unsigned char chr, unsigned char *_31F4x)
{
	unsigned char	bchr, c;
	const unsigned char bmask_31f4[2] = { 0x00, 0x84 };
	const unsigned char bmask_31f5[2] = { 0x00, 0x08 };
	unsigned char wk31f4(_31F4x[0]), wk31f5(_31F4x[1]), wk31fa;

	bchr = bitrev(chr);
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	_31F4x[0] = wk31f4;	_31F4x[1] = wk31f5;		// 4, 5

	c = (wk31f4 >= 0xE5) ? 1 : 0;
	_31F4x[2] = adc(c, chr, _31F4x[2]);			// 7

	_31F4x[3] = adc(c, _31F4x[3], _31F4x[1]);	// 8

	_31F4x[4] = chr ^ _31F4x[4];				// 9

	wk31fa = ror(c, _31F4x[5]);
	_31F4x[5] = adc(c, wk31fa, chr);			// A

	_31F4x[6] += c + bitcnt(chr);				// B

	return (c);
}

// <<< dim3(64 * 64, 64), dim3(64) >>>
__global__ void calc_1_4col(
	unsigned char *work4col,			// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	unsigned char *valid,				// �L���ȕ����R�[�h�ō\�����ꂽ��
	const unsigned char *chrmask,		// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const unsigned char *lut_x31F4,		// $31F4���o�p��LUT
	const unsigned char *lut_x31F5		// $31F5���o�p��LUT
)
{
	const unsigned int idx = ((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x;
	const unsigned int gblidx = idx * 8;

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;

	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	/*$31F4*/	work4col[gblidx + 0] = 0;
	/*$31F5*/	work4col[gblidx + 1] = 0;
	/*$31F7*/	work4col[gblidx + 2] = 0;
	/*$31F8*/	work4col[gblidx + 3] = 0;
	/*$31F9*/	work4col[gblidx + 4] = 0;
	/*$31FA*/	work4col[gblidx + 5] = 1;
	/*$31FB*/	work4col[gblidx + 6] = 0;

	calcstep(chr1, &work4col[gblidx]);
	calcstep(chr2, &work4col[gblidx]);
	calcstep(chr3, &work4col[gblidx]);
	calcstep(chr4, &work4col[gblidx]);

	/*carry*/	work4col[gblidx + 7] = 0;
				// F6�͎����Œ�l�Ȃ̂ł߂Ď��̌��ɓn���L�����[���i�[����

	valid[idx] = chrmask[chr1] | chrmask[chr2] | chrmask[chr3] | chrmask[chr4];
}

// ����܂ł̌��̒l���g���Ēǉ���4�����̌v�Z�l�����߂�
__global__ void calc_4col(
	unsigned char		*work8col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	unsigned char		*unvalid8,		// �����������܂܂�Ă�����true
	const unsigned char	*work4col,		// ����܂łɌv�Z���Ă������Z����
	const unsigned char *chrmask,		// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const unsigned char *lut_x31F4,		// $31F4���o�p��LUT
	const unsigned char *lut_x31F5,		// $31F5���o�p��LUT
	const unsigned int	idx4			// work4col�ɑΉ�����C���f�b�N�X�l
)
{
	// �X���b�h�ԍ����珑�����ݐ�C���f�b�N�X���v�Z
	const unsigned int idx = ((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x;
	const unsigned int gblidx = idx * 8;

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;
	const unsigned int	rdidx = idx4 * 8;

	/*$31F4*/	work8col[gblidx + 0] = work4col[rdidx + 0];
	/*$31F5*/	work8col[gblidx + 1] = work4col[rdidx + 1];
	/*$31F7*/	work8col[gblidx + 2] = work4col[rdidx + 2];
	/*$31F8*/	work8col[gblidx + 3] = work4col[rdidx + 3];
	/*$31F9*/	work8col[gblidx + 4] = work4col[rdidx + 4];
	/*$31FA*/	work8col[gblidx + 5] = work4col[rdidx + 5];
	/*$31FB*/	work8col[gblidx + 6] = work4col[rdidx + 6];

	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	calcstep(chr1, &work8col[gblidx]);
	calcstep(chr2, &work8col[gblidx]);
	calcstep(chr3, &work8col[gblidx]);
	calcstep(chr4, &work8col[gblidx]);

	/*carry*/	work8col[gblidx + 7] = 0;
	// F6�͎����Œ�l�Ȃ̂ł߂Ď��̌��ɓn���L�����[���i�[����

	unvalid8[idx] = chrmask[chr1] | chrmask[chr2] | chrmask[chr3] | chrmask[chr4];
}

// �c��̂P�����o�ƍŏI���܂ł̃`�F�b�N�f�B�W�b�g�v�Z�A�L������
__global__ void calclast_validate(
	unsigned char		*result,	// OUT  14�����̊e�f�[�^�v�Z����
	unsigned char		*valid,		// OUT  �p�X���[�h�Ƃ��Đ��藧�Ȃ�1(true)�łȂ����0(false)
	const unsigned char	*work,		// IN	12���ڂ܂łœ��o���ꂽ�r���l
	const unsigned char *lut_x31F4,	// IN	$31F4���o�p��LUT
	const unsigned char *lut_x31F5,	// IN	$31F5���o�p��LUT
	const unsigned char	*chrmask,	// IN	�L���ȕ����R�[�h���ǂ����̃t���O�}�b�v
	const unsigned int	workidx		// IN	blockIdx.x�̃I�t�Z�b�g
)
{
	const unsigned int	wkidx = workidx * 8;
	const unsigned int	gblidx = ((blockIdx.x * blockDim.x) + threadIdx.x * 8);
	const unsigned char col13 = blockIdx.x;			// �u���b�Nidx.x��13���ڂ̕����R�[�h���
	const unsigned char col14 = threadIdx.x;		// �X���b�hidx.x��14���ڂ̕����R�[�h���

	/*$31F4*/	result[gblidx + 0] = work[wkidx + 0];
	/*$31F5*/	result[gblidx + 1] = work[wkidx + 1];
	/*$31F7*/	result[gblidx + 2] = work[wkidx + 2];
	/*$31F8*/	result[gblidx + 3] = work[wkidx + 3];
	/*$31F9*/	result[gblidx + 4] = work[wkidx + 4];
	/*$31FA*/	result[gblidx + 5] = work[wkidx + 5];
	/*$31FB*/	result[gblidx + 6] = work[wkidx + 6];

	const unsigned char chr13(col13 & 0x3F), chr14(col14 & 0x3F);

	calcstep(chr13, &result[gblidx]);
	calcstep(chr14, &result[gblidx]);

	bool	judge = true
		&& (result[gblidx + 0] == 0x65)	// $31F4
		&& (result[gblidx + 1] == 0x94)	// $31F5
		&& (result[gblidx + 2] == 0xAC)	// $31F7
		&& (result[gblidx + 3] == 0xE9)	// $31F8
		&& (result[gblidx + 4] == 0x07)	// $31F9
		&& (result[gblidx + 5] == 0x33)	// $31FA
		&& (result[gblidx + 6] == 0x25)	// $31FB
		;

	/*Passed*/	result[gblidx + 7] = judge;

	valid[(blockIdx.x * blockDim.x) + threadIdx.x] = chrmask[chr13] | chrmask[chr14];
}

const unsigned char lut_xor31F4[256] = {
	0x00,0x11,0x23,0x32,0x46,0x57,0x65,0x74,0x8C,0x9D,0xAF,0xBE,0xCA,0xDB,0xE9,0xF8,
	0x10,0x01,0x33,0x22,0x56,0x47,0x75,0x64,0x9C,0x8D,0xBF,0xAE,0xDA,0xCB,0xF9,0xE8,
	0x21,0x30,0x02,0x13,0x67,0x76,0x44,0x55,0xAD,0xBC,0x8E,0x9F,0xEB,0xFA,0xC8,0xD9,
	0x31,0x20,0x12,0x03,0x77,0x66,0x54,0x45,0xBD,0xAC,0x9E,0x8F,0xFB,0xEA,0xD8,0xC9,
	0x42,0x53,0x61,0x70,0x04,0x15,0x27,0x36,0xCE,0xDF,0xED,0xFC,0x88,0x99,0xAB,0xBA,
	0x52,0x43,0x71,0x60,0x14,0x05,0x37,0x26,0xDE,0xCF,0xFD,0xEC,0x98,0x89,0xBB,0xAA,
	0x63,0x72,0x40,0x51,0x25,0x34,0x06,0x17,0xEF,0xFE,0xCC,0xDD,0xA9,0xB8,0x8A,0x9B,
	0x73,0x62,0x50,0x41,0x35,0x24,0x16,0x07,0xFF,0xEE,0xDC,0xCD,0xB9,0xA8,0x9A,0x8B,
	0x84,0x95,0xA7,0xB6,0xC2,0xD3,0xE1,0xF0,0x08,0x19,0x2B,0x3A,0x4E,0x5F,0x6D,0x7C,
	0x94,0x85,0xB7,0xA6,0xD2,0xC3,0xF1,0xE0,0x18,0x09,0x3B,0x2A,0x5E,0x4F,0x7D,0x6C,
	0xA5,0xB4,0x86,0x97,0xE3,0xF2,0xC0,0xD1,0x29,0x38,0x0A,0x1B,0x6F,0x7E,0x4C,0x5D,
	0xB5,0xA4,0x96,0x87,0xF3,0xE2,0xD0,0xC1,0x39,0x28,0x1A,0x0B,0x7F,0x6E,0x5C,0x4D,
	0xC6,0xD7,0xE5,0xF4,0x80,0x91,0xA3,0xB2,0x4A,0x5B,0x69,0x78,0x0C,0x1D,0x2F,0x3E,
	0xD6,0xC7,0xF5,0xE4,0x90,0x81,0xB3,0xA2,0x5A,0x4B,0x79,0x68,0x1C,0x0D,0x3F,0x2E,
	0xE7,0xF6,0xC4,0xD5,0xA1,0xB0,0x82,0x93,0x6B,0x7A,0x48,0x59,0x2D,0x3C,0x0E,0x1F,
	0xF7,0xE6,0xD4,0xC5,0xB1,0xA0,0x92,0x83,0x7B,0x6A,0x58,0x49,0x3D,0x2C,0x1E,0x0F,
};

const unsigned char lut_xor31F5[256] = {
	0x00,0x89,0x12,0x9B,0x24,0xAD,0x36,0xBF,0x48,0xC1,0x5A,0xD3,0x6C,0xE5,0x7E,0xF7,
	0x81,0x08,0x93,0x1A,0xA5,0x2C,0xB7,0x3E,0xC9,0x40,0xDB,0x52,0xED,0x64,0xFF,0x76,
	0x02,0x8B,0x10,0x99,0x26,0xAF,0x34,0xBD,0x4A,0xC3,0x58,0xD1,0x6E,0xE7,0x7C,0xF5,
	0x83,0x0A,0x91,0x18,0xA7,0x2E,0xB5,0x3C,0xCB,0x42,0xD9,0x50,0xEF,0x66,0xFD,0x74,
	0x04,0x8D,0x16,0x9F,0x20,0xA9,0x32,0xBB,0x4C,0xC5,0x5E,0xD7,0x68,0xE1,0x7A,0xF3,
	0x85,0x0C,0x97,0x1E,0xA1,0x28,0xB3,0x3A,0xCD,0x44,0xDF,0x56,0xE9,0x60,0xFB,0x72,
	0x06,0x8F,0x14,0x9D,0x22,0xAB,0x30,0xB9,0x4E,0xC7,0x5C,0xD5,0x6A,0xE3,0x78,0xF1,
	0x87,0x0E,0x95,0x1C,0xA3,0x2A,0xB1,0x38,0xCF,0x46,0xDD,0x54,0xEB,0x62,0xF9,0x70,
	0x08,0x81,0x1A,0x93,0x2C,0xA5,0x3E,0xB7,0x40,0xC9,0x52,0xDB,0x64,0xED,0x76,0xFF,
	0x89,0x00,0x9B,0x12,0xAD,0x24,0xBF,0x36,0xC1,0x48,0xD3,0x5A,0xE5,0x6C,0xF7,0x7E,
	0x0A,0x83,0x18,0x91,0x2E,0xA7,0x3C,0xB5,0x42,0xCB,0x50,0xD9,0x66,0xEF,0x74,0xFD,
	0x8B,0x02,0x99,0x10,0xAF,0x26,0xBD,0x34,0xC3,0x4A,0xD1,0x58,0xE7,0x6E,0xF5,0x7C,
	0x0C,0x85,0x1E,0x97,0x28,0xA1,0x3A,0xB3,0x44,0xCD,0x56,0xDF,0x60,0xE9,0x72,0xFB,
	0x8D,0x04,0x9F,0x16,0xA9,0x20,0xBB,0x32,0xC5,0x4C,0xD7,0x5E,0xE1,0x68,0xF3,0x7A,
	0x0E,0x87,0x1C,0x95,0x2A,0xA3,0x38,0xB1,0x46,0xCF,0x54,0xDD,0x62,0xEB,0x70,0xF9,
	0x8F,0x06,0x9D,0x14,0xAB,0x22,0xB9,0x30,0xC7,0x4E,0xD5,0x5C,0xE3,0x6A,0xF1,0x78,
};

using namespace std;

// const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!.-nmc�Ѻ" };
const unsigned char charvalidmask[64] = {
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,	// 0x06��0x07������
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
};

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
);

const unsigned long	WORKSIZE = 64 * 64 * 64 * 64;
static unsigned char result[WORKSIZE];

int main()
{
	hipError_t cudaStatus;

	chkthread(result, WORKSIZE);

	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	return 0;
}

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
)
{
	hipError_t cudaStatus;
	unsigned char *dev_chrcode_mask = 0;	// �L�������R�[�h����p�}�X�N		�L����0x00�A������0x80�B
	unsigned char *dev_validpass = 0;		// �v�Z�������Ȃ��1������

	unsigned char *dev_31F4_xortbl = 0;		// $31F4���o�p�e�[�u��
	unsigned char *dev_31F5_xortbl = 0;		// $31F5���o�p�e�[�u��

	unsigned char *cpu_validpass = 0;		// �L���p�X���[�h���茋��(CPU��)
	unsigned char *cpu_calcresult = 0;		// �v�Z���ʊm�F�p

	unsigned char *cpu_unvalid4 = 0;		// �L�������ō\������Ă��邩���(�擪4��)
	unsigned char *cpu_unvalid8 = 0;		// �L�������ō\������Ă��邩���(5-8��)
	unsigned char *cpu_unvalid12 = 0;		// �L�������ō\������Ă��邩���(9-12��)

	unsigned char *dev_unvalid4 = 0;		// �L�������ō\������Ă��邩���(�擪4��)
	unsigned char *dev_unvalid8 = 0;		// �L�������ō\������Ă��邩���(5-8��)
	unsigned char *dev_unvalid12 = 0;		// �L�������ō\������Ă��邩���(9-12��)

	unsigned char *dev_res4 = 0;			// �擪4���Ԃ�̌v�Z���ʂƃL�����[���
	unsigned char *dev_res8 = 0;			// 5-8���܂ł̌v�Z���ʂƃL�����[���
	unsigned char *dev_res12 = 0;			// 9-12���܂Ōv�Z���ʂƃL�����[���
	unsigned char *dev_result = 0;			// 14���̌v�Z����

	unsigned long long validcnt = 0;		// �`�F�b�N�f�B�W�b�g��ʂ����p�X���[�h�̌�

	dim3	 block_1677m(64), grid_1677m(64 * 64, 64);	// ����64�� 4�����̕���v�Z�O���b�h�p	16.8M items

	cudaStatus = hipSetDevice(0);		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	
	// ���o�p�e�[�u��(const)
	cudaStatus = hipMalloc((void**)&dev_chrcode_mask, 64 * sizeof(unsigned char));						if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_31F4_xortbl, 256 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_31F5_xortbl, 256 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 4�����̊e�f�[�^�Z�o�p�̈�m��
	cudaStatus = hipMalloc((void**)&dev_res4, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res8, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res12, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_result, 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_validpass, 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_unvalid4, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_unvalid8, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_unvalid12, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }


	cudaStatus = hipMemcpy(dev_chrcode_mask, charvalidmask, 64 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_31F4_xortbl, lut_xor31F4, 256 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_31F5_xortbl, lut_xor31F5, 256 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_validpass, 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_calcresult, 64 * 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_unvalid4, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_unvalid8, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_unvalid12, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �擪�S���S�g�݂��킹�̌v�Z
	calc_1_4col <<< dim3(64 * 64, 64), dim3(64) >>> (dev_res4, dev_unvalid4, dev_chrcode_mask, dev_31F4_xortbl, dev_31F5_xortbl);	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calu14col!\n", cudaStatus); goto Error; }
	cudaStatus = hipMemcpy(cpu_unvalid4, dev_unvalid4, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

	//cudaStatus = hipMemcpy(cpu_calcresult, dev_res4, (64 * 64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
	//for (int chkrs = 0; chkrs < 16/*(64 * 64 * 64)*/; ++chkrs) {
	//	printf("\n");

	//	for (int vidx = 0; vidx < 8; ++vidx) {
	//		printf("%02x ", cpu_calcresult[chkrs * 8 + vidx]);
	//	}
	//}

	// �\���������茋�ʎ��o��

	validcnt = 0;

	for (int xor4idx = 0; xor4idx < (64 * 64 * 64 * 64); ++xor4idx) {
		// �����������܂܂�Ă����猟���Ώۏ��O
		if (cpu_unvalid4[xor4idx]) {
			continue;
		}
		
		// 5-8���̑g�������v�Z����
		calc_4col << < dim3(64 * 64, 64), dim3(64) >> > (dev_res8, dev_unvalid8, dev_res4, dev_chrcode_mask, dev_31F4_xortbl, dev_31F5_xortbl, xor4idx);
		cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
		cudaStatus = hipMemcpy(cpu_unvalid8, dev_unvalid8, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

		//cudaStatus = hipMemcpy(cpu_calcresult, dev_res8, (64 * 64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
		//for (int chkrs = 0; chkrs < (64 * 64 * 64); ++chkrs) {
		//	printf("\n");

		//	for (int vidx = 0; vidx < 8; ++vidx) {
		//		printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
		//	}
		//}

		for (int xor8idx = 0; xor8idx < (64 * 64 * 64 * 64); ++xor8idx) {
			if (cpu_unvalid8[xor8idx]) {
				continue;
			}
			// 9�`12�����̑g�������v�Z����
			calc_4col << < dim3(64 * 64, 64), 64 >> > (dev_res12, dev_unvalid12, dev_res8, dev_chrcode_mask, dev_31F4_xortbl, dev_31F5_xortbl, xor8idx);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
			cudaStatus = hipMemcpy(cpu_unvalid12, dev_unvalid12, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

			//cudaStatus = hipMemcpy(cpu_calcresult, dev_res12, (64 * 64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
			//for (int chkrs = 0; chkrs < (64 * 64 * 64); ++chkrs) {
			//	printf("\n");

			//	for (int vidx = 0; vidx < 8; ++vidx) {
			//		printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
			//	}
			//}

			for (int xor12idx = 0; xor12idx < (64 * 64 * 64 * 64); ++xor12idx) {
				if (cpu_unvalid12[xor12idx]) {
					continue;
				}
				// 13,14���ڂ̓��o�ƃ`�F�b�N�f�B�W�b�g���ʂ��Ă��邩�̔���
				calclast_validate << < 64, 64 >> > (dev_result, dev_validpass, dev_res12, dev_31F4_xortbl, dev_31F5_xortbl, dev_chrcode_mask, xor12idx);
				cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching validation!\n", cudaStatus); goto Error; }


				cudaStatus = hipMemcpy(cpu_validpass, dev_validpass, 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
				cudaStatus = hipMemcpy(cpu_calcresult, dev_result, (64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
				for (int chkrs = 0; chkrs < 16/*(64 * 64 * 64)*/; ++chkrs) {
//					if (!cpu_calcresult[chkrs * 8 + 7] ) continue;
					printf("\n");

					for (int vidx = 0; vidx < 8; ++vidx) {
						printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
					}
					printf("| %02X ", cpu_validpass[chkrs]);
					validcnt += cpu_calcresult[chkrs * 8 + 7];
				}
				goto FIN;
				printf("\n%lld items\n", validcnt);


			}
		}
	}
FIN:
	printf("found %lld items\n", validcnt);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(cpu_result, dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }


Error:
	hipFree(dev_chrcode_mask);
	hipFree(dev_validpass);

	hipFree(dev_31F4_xortbl);
	hipFree(dev_31F5_xortbl);

	hipFree(cpu_validpass);
	hipFree(cpu_calcresult);

	hipFree(dev_unvalid4);
	hipFree(dev_unvalid8);
	hipFree(dev_unvalid12);

	hipFree(dev_res4);
	hipFree(dev_res8);
	hipFree(dev_res12);
	hipFree(dev_result);

	hipFree(dev_unvalid4);
	hipFree(dev_unvalid8);
	hipFree(dev_unvalid12);

	return(cudaStatus);

}
