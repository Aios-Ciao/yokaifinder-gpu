#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <stdio.h>

#define PASSWORD_LEN		(14)
#define PASSWORD_LEN_MAX	(16)

// �p�X���[�h�̌v�Z�J�[�l��
__global__ void checkPassKernel(
	unsigned char *result,			// OUTPUT	�p�X������TRUE��Ԃ���	[x*y]
	const unsigned char*candidate,	// INPUT	�p�X���[�h���(�����R�[�h�ϊ��ς�) [x*y][16]
	const unsigned char*refer_chk	// INPUT	����p�`�F�b�N�f�B�W�b�g	[x*y][16]
)
{
	bool result_b(false), match_flg;
	int a(0), c(0), cb(0), chridx, strlength;
	unsigned char mem20(0), mem21(0);
	unsigned char mem50(0), mem51(0), mem52(0), mem53(1), mem54(0);

	int th_x = threadIdx.x;		// 
//	int th_y = threadIdx.y;

	auto ror = [&c](unsigned char v)
	{
		unsigned char wc = c;

		c = v & 0x01;
		return (unsigned char)((v >> 1) | (wc << 7));
	};
	auto adc = [&c](unsigned char lhs, unsigned char rhs)
	{
		unsigned short v;

		v = (unsigned short)lhs + (unsigned short)rhs + (unsigned short)c;
		c = (v > 0xFF) ? 1 : 0;

		return (v & 0xFF);
	};
	auto bitcnt = [](unsigned long int v)
	{
			v = (v & 0x55555555) + (v >> 1 & 0x55555555);
			v = (v & 0x33333333) + (v >> 2 & 0x33333333);
			v = (v & 0x0f0f0f0f) + (v >> 4 & 0x0f0f0f0f);
			v = (v & 0x00ff00ff) + (v >> 8 & 0x00ff00ff);
		return  (v & 0x0000ffff) + (v >> 16 & 0x0000ffff);
	};

	strlength = refer_chk[2];
	for (chridx = 0; chridx < strlength; ++chridx) {
		a = candidate[(th_x * PASSWORD_LEN_MAX) + chridx];

		// calc checkdigit1	(���[�v�W�J���Ă���)
//		for (int bit = 7; bit >= 0; --bit) {
//			c = (a >> bit) & 1;
//			mem20 = ror(mem20);
//			mem21 = ror(mem21);
//
//			if (c) {
//				mem20 ^= 0x84;
//				mem21 ^= 0x08;
//			}
//		}
		c = ((a & 0x80) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x40) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x20) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x10) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x08) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x04) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x02) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x01) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		// calc checkdigit2
		c = (mem20 >= 0xE5) ? 1 : 0;
		mem50 = adc(a, mem50);
		mem51 = adc(mem51, mem21);
		// calc checkdigit3
		mem52 ^= a;
		// calc checkdigit4
		{
			unsigned char v = ror(mem53);
			mem53 = adc(v, a);
		}
		// calc checkdigit5
		mem54 += (unsigned char)(c + bitcnt(a));
	}
	
	result_b = true;
	match_flg = (mem20 == refer_chk[0]);
	result_b = result_b && match_flg;
	match_flg = (mem21 == refer_chk[1]);
	result_b = result_b && match_flg;
	match_flg = (mem50 == refer_chk[3]);
	result_b = result_b && match_flg;
	match_flg = (mem51 == refer_chk[4]);
	result_b = result_b && match_flg;
	match_flg = (mem52 == refer_chk[5]);
	result_b = result_b && match_flg;
	match_flg = (mem53 == refer_chk[6]);
	result_b = result_b && match_flg;
	match_flg = (mem54 == refer_chk[7]);
	result_b = result_b && match_flg;

	result[th_x] = result_b;
}


// 2�����̃p�X���[�h���𐶐�����
__global__ void chrconvKernel(
	unsigned char *candidate,			// OUTPUT	����������f�[�^
	const unsigned char *basecode,		// INPUT	�Œ蕔���Ƃ��ĎQ�Ƃ��錅���[16]
	const unsigned char *tbl_chrdict,	// INPUT	�����R�[�h�e�[�u��[256]
	const unsigned int	passlen,		// INPUT	�ϊ��Ώۂ̌���(�S��)
	const unsigned int	dictlen			// INPUT	�����퐔
)
{
	int th_x = threadIdx.x;

	for (int i = 0; i < (passlen - 1); ++i) {
		candidate[(PASSWORD_LEN_MAX * th_x) + i] = tbl_chrdict[basecode[i]];
	}
}

__global__ void make_4col_fullpair(
	unsigned char *pass4col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*4����͂��܂�4byte
	unsigned char *chrmask,			// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const int  offset				// blockIdx��z�ɗ������鉺��
)
{
	unsigned int	blocksize = blockDim.x * blockDim.y * blockDim.z;
	unsigned int	thidx = (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int	gridx = ((gridDim.x * gridDim.y) * ((blockIdx.z + (offset*gridDim.z)))) + (gridDim.x * blockIdx.y) + blockIdx.x;
	unsigned int	gblidx = gridx * blocksize + thidx;

	unsigned char	col1 = blockIdx.z + (offset * gridDim.z);
	unsigned char	col2 = blockIdx.y;
	unsigned char	col3 = blockIdx.x * (blockDim.z * blockDim.y) + threadIdx.z * blockDim.y + threadIdx.y;
	unsigned char	col4 = threadIdx.x;
	
	unsigned char unvalid;

	unvalid = chrmask[col1] | chrmask[col2] | chrmask[col3] | chrmask[col4];

	pass4col[gblidx] = unvalid | (col1 ^ col2 ^ col3 ^ col4);
}

// ����܂ł�XOR���ʂ�4���Ԃ��XOR�����߂�
__global__ void calcxor8(
	unsigned char		*xor8,		// ���Z����
	const unsigned char *xor4,		// ����܂ł̉��Z����
	const unsigned int	idxG		// xor8�̃C���f�b�N�X
)
{
	unsigned int idxH = (blockIdx.x * 256) + threadIdx.x;

	unsigned char W1 = xor4[idxG];
	unsigned char W2 = xor4[idxH];
	unsigned char enf = (W1 | W2) & 0x80;

	xor8[idxH] = enf | (W1 ^ W2);
}

// 13���ڂ̓��o��xor 0x07���ʂ̑Ó������f
__global__ void calcxor13_validchk(
	unsigned char		*valid14,		// 14���ڂ��L���Ȃ�1���i�[
	const unsigned char *xor12,			// 12���ڂ܂œ��o���ꂽXOR
	const unsigned char *chrmask		// �L���ȕ����R�[�h���ǂ���
)
{
	unsigned char col13 = xor12[blockIdx.x] ^ threadIdx.x;	// 13���ڂ̓��o 1677���v�f�̓��Y���l

	col13 |= chrmask[threadIdx.x];
	col13 ^= 0x07;

	valid14[(blockIdx.x*blockDim.x) + threadIdx.x] = !(col13 & 0x80);	// �����ȕ����R�[�h�������������
}

// �J�[�l���Ăяo���e�X�g�p
__global__ void kerntest()
{
}

const unsigned char lut_xor31F4[256] = {
	0x00,0x11,0x23,0x32,0x46,0x57,0x65,0x74,0x8C,0x9D,0xAF,0xBE,0xCA,0xDB,0xE9,0xF8,
	0x10,0x01,0x33,0x22,0x56,0x47,0x75,0x64,0x9C,0x8D,0xBF,0xAE,0xDA,0xCB,0xF9,0xE8,
	0x21,0x30,0x02,0x13,0x67,0x76,0x44,0x55,0xAD,0xBC,0x8E,0x9F,0xEB,0xFA,0xC8,0xD9,
	0x31,0x20,0x12,0x03,0x77,0x66,0x54,0x45,0xBD,0xAC,0x9E,0x8F,0xFB,0xEA,0xD8,0xC9,
	0x42,0x53,0x61,0x70,0x04,0x15,0x27,0x36,0xCE,0xDF,0xED,0xFC,0x88,0x99,0xAB,0xBA,
	0x52,0x43,0x71,0x60,0x14,0x05,0x37,0x26,0xDE,0xCF,0xFD,0xEC,0x98,0x89,0xBB,0xAA,
	0x63,0x72,0x40,0x51,0x25,0x34,0x06,0x17,0xEF,0xFE,0xCC,0xDD,0xA9,0xB8,0x8A,0x9B,
	0x73,0x62,0x50,0x41,0x35,0x24,0x16,0x07,0xFF,0xEE,0xDC,0xCD,0xB9,0xA8,0x9A,0x8B,
	0x84,0x95,0xA7,0xB6,0xC2,0xD3,0xE1,0xF0,0x08,0x19,0x2B,0x3A,0x4E,0x5F,0x6D,0x7C,
	0x94,0x85,0xB7,0xA6,0xD2,0xC3,0xF1,0xE0,0x18,0x09,0x3B,0x2A,0x5E,0x4F,0x7D,0x6C,
	0xA5,0xB4,0x86,0x97,0xE3,0xF2,0xC0,0xD1,0x29,0x38,0x0A,0x1B,0x6F,0x7E,0x4C,0x5D,
	0xB5,0xA4,0x96,0x87,0xF3,0xE2,0xD0,0xC1,0x39,0x28,0x1A,0x0B,0x7F,0x6E,0x5C,0x4D,
	0xC6,0xD7,0xE5,0xF4,0x80,0x91,0xA3,0xB2,0x4A,0x5B,0x69,0x78,0x0C,0x1D,0x2F,0x3E,
	0xD6,0xC7,0xF5,0xE4,0x90,0x81,0xB3,0xA2,0x5A,0x4B,0x79,0x68,0x1C,0x0D,0x3F,0x2E,
	0xE7,0xF6,0xC4,0xD5,0xA1,0xB0,0x82,0x93,0x6B,0x7A,0x48,0x59,0x2D,0x3C,0x0E,0x1F,
	0xF7,0xE6,0xD4,0xC5,0xB1,0xA0,0x92,0x83,0x7B,0x6A,0x58,0x49,0x3D,0x2C,0x1E,0x0F,
};

const unsigned char lut_xor31F5[256] = {
	0x00,0x89,0x12,0x9B,0x24,0xAD,0x36,0xBF,0x48,0xC1,0x5A,0xD3,0x6C,0xE5,0x7E,0xF7,
	0x81,0x08,0x93,0x1A,0xA5,0x2C,0xB7,0x3E,0xC9,0x40,0xDB,0x52,0xED,0x64,0xFF,0x76,
	0x02,0x8B,0x10,0x99,0x26,0xAF,0x34,0xBD,0x4A,0xC3,0x58,0xD1,0x6E,0xE7,0x7C,0xF5,
	0x83,0x0A,0x91,0x18,0xA7,0x2E,0xB5,0x3C,0xCB,0x42,0xD9,0x50,0xEF,0x66,0xFD,0x74,
	0x04,0x8D,0x16,0x9F,0x20,0xA9,0x32,0xBB,0x4C,0xC5,0x5E,0xD7,0x68,0xE1,0x7A,0xF3,
	0x85,0x0C,0x97,0x1E,0xA1,0x28,0xB3,0x3A,0xCD,0x44,0xDF,0x56,0xE9,0x60,0xFB,0x72,
	0x06,0x8F,0x14,0x9D,0x22,0xAB,0x30,0xB9,0x4E,0xC7,0x5C,0xD5,0x6A,0xE3,0x78,0xF1,
	0x87,0x0E,0x95,0x1C,0xA3,0x2A,0xB1,0x38,0xCF,0x46,0xDD,0x54,0xEB,0x62,0xF9,0x70,
	0x08,0x81,0x1A,0x93,0x2C,0xA5,0x3E,0xB7,0x40,0xC9,0x52,0xDB,0x64,0xED,0x76,0xFF,
	0x89,0x00,0x9B,0x12,0xAD,0x24,0xBF,0x36,0xC1,0x48,0xD3,0x5A,0xE5,0x6C,0xF7,0x7E,
	0x0A,0x83,0x18,0x91,0x2E,0xA7,0x3C,0xB5,0x42,0xCB,0x50,0xD9,0x66,0xEF,0x74,0xFD,
	0x8B,0x02,0x99,0x10,0xAF,0x26,0xBD,0x34,0xC3,0x4A,0xD1,0x58,0xE7,0x6E,0xF5,0x7C,
	0x0C,0x85,0x1E,0x97,0x28,0xA1,0x3A,0xB3,0x44,0xCD,0x56,0xDF,0x60,0xE9,0x72,0xFB,
	0x8D,0x04,0x9F,0x16,0xA9,0x20,0xBB,0x32,0xC5,0x4C,0xD7,0x5E,0xE1,0x68,0xF3,0x7A,
	0x0E,0x87,0x1C,0x95,0x2A,0xA3,0x38,0xB1,0x46,0xCF,0x54,0xDD,0x62,0xEB,0x70,0xF9,
	0x8F,0x06,0x9D,0x14,0xAB,0x22,0xB9,0x30,0xC7,0x4E,0xD5,0x5C,0xE3,0x6A,0xF1,0x78,
};

// �����̃r�b�g���㉺���]����e�[�u��
unsigned char lut_reverse_char_bit[] = {
  0b00000000, 0b10000000, 0b01000000, 0b11000000, 0b00100000, 0b10100000, 0b01100000, 0b11100000,	// 00 - 07
  0b00010000, 0b10010000, 0b01010000, 0b11010000, 0b00110000, 0b10110000, 0b01110000, 0b11110000,	// 08 - 0F
  0b00001000, 0b10001000, 0b01001000, 0b11001000, 0b00101000, 0b10101000, 0b01101000, 0b11101000,	// 10 - 17
  0b00011000, 0b10011000, 0b01011000, 0b11011000, 0b00111000, 0b10111000, 0b01111000, 0b11111000,	// 18 - 1F
  0b00000100, 0b10000100, 0b01000100, 0b11000100, 0b00100100, 0b10100100, 0b01100100, 0b11100100,	// 20 - 27
  0b00010100, 0b10010100, 0b01010100, 0b11010100, 0b00110100, 0b10110100, 0b01110100, 0b11110100,	// 28 - 2F
  0b00001100, 0b10001100, 0b01001100, 0b11001100, 0b00101100, 0b10101100, 0b01101100, 0b11101100,	// 30 - 37
  0b00011100, 0b10011100, 0b01011100, 0b11011100, 0b00111100, 0b10111100, 0b01111100, 0b11111100,	// 38 - 3F
};

using namespace std;

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t chkPass(unsigned char *, string, int, unsigned char *);

const unsigned char g_chrcode[256] =
{
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x2D,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x2B,0x00,0x00,
	0x25,0x04,0x0C,0x14,0x1C,0x24,0x05,0x0D,0x15,0x1D,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x08,0x10,0x18,0x20,0x28,0x30,0x01,0x09,0x11,0x19,0x21,0x29,0x31,0x02,
	0x0A,0x12,0x1A,0x22,0x2A,0x32,0x03,0x0B,0x13,0x1B,0x23,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x33,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x35,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x2C,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x34,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
	0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,0x00,
};

//const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ" };
const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!.-nmc�Ѻ" };

const unsigned char charvalidmask[64] = { 
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,	// 0x06��0x07������
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
};

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
);

const unsigned long	WORKSIZE = 64 * 64 * 64 * 64;
static unsigned char result[WORKSIZE];

int main()
{
	chkthread(result, WORKSIZE);

#if 0
	unsigned char	pass_columns[PASSWORD_LEN_MAX] = { 0 };
	int passed = -1;

	hipError_t cudaStatus;

	const size_t	dictlen = sizeof(_dict);
	std::string chrtable(_dict, dictlen);
	unsigned char *chkresult = new unsigned char[dictlen * dictlen];

	const int		COLUMNS = 14;
	const int		refresh_col_n = 4;
	const int		chkcol = 8;
	unsigned char	chkkey;
	unsigned char	refresh_col;		// �i���\���p�̔��茅

	// �I������p���l��ޔ�
	chkkey = pass_columns[chkcol];
	refresh_col = pass_columns[refresh_col_n];

	while (pass_columns[chkcol] == chkkey) {

		if (refresh_col != pass_columns[refresh_col_n]) {

			for (int idx = 0; idx < PASSWORD_LEN; ++idx) {
				fprintf(stderr, "%c", _dict[pass_columns[idx]]);
			}
			fprintf(stderr, "\n");
			refresh_col = pass_columns[refresh_col_n];
		}

		hipError_t cudaStatus = chkPass(chkresult, chrtable, 14, pass_columns);
		if (cudaStatus != hipSuccess) { fprintf(stderr, "passcheck failed!"); return 1; }

		for (int idx = 0; idx < (chrtable.size() * chrtable.size()); ++idx) {
			if (chkresult[idx] != 0) {
				fprintf(stderr, "Passed %d\n", idx);
			}
		}

		// �I�������ɍ��v�����Ԃ𔻒肵����T���I��
		bool carry = true;
		for (int col = (2); col < (COLUMNS - 1); ++col) {
			//	for (int col = 2; col < (COLUMNS - 1); ++col) {
			pass_columns[col] += carry ? 1 : 0;
			carry = (pass_columns[col] == dictlen);
			pass_columns[col] = carry ? 0 : pass_columns[col];
			//	candidate[col] = dict[counters[col]];
		}
	}

	
	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	delete[] chkresult;
#endif
	return 0;
}

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
)
{
	hipError_t cudaStatus;
//	unsigned char *dev_result = 0;		// ���茋��
	unsigned char *dev_xor_result = 0;		// 4���Ԃ��xor����
	unsigned char *dev_xor_result8 = 0;		// 8���Ԃ��xor����(�e���|����)
	unsigned char *dev_xor_result12 = 0;	// 12���Ԃ��xor����(�e���|����)
	unsigned char *dev_31F9_is07h = 0;		// 13���ڂ܂�xor�����߂����ʂ�0x07��xor�����l(14���ڂ̕����R�[�h)���L���ȕ����R�[�h��

	unsigned char *dev_chrcode_mask = 0;	// �L�������R�[�h����p�}�X�N		�L����0x00�A������0x80�B

	unsigned char *cpu_31F9_valid = 0;
	unsigned long validcnt = 0;


//	dim3	 block_530m(48, 8, 2), grid_530m(3, 48, 6);	// ����48�� 4�����̕���v�Z�O���b�h�p	 5.3M items
	dim3	 block_1677m(64, 8, 2), grid_1677m(4, 64, 4);	// ����64�� 4�����̕���v�Z�O���b�h�p	16.8M items

	cudaStatus = hipSetDevice(0);																		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	// 4�������g������XOR�f�[�^����
	cudaStatus = hipMalloc((void**)&dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_xor_result8, 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_xor_result12, 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_31F9_is07h, 64 * 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 64�����핪�̃R�[�h�L���������
	cudaStatus = hipMalloc((void**)&dev_chrcode_mask, 64 * sizeof(unsigned char));						if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_chrcode_mask, charvalidmask, 64 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_31F9_valid, 64 * 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 4�������g������XOR�f�[�^�v�Z
	for (int offset = 0; offset < 16; ++offset) {
		make_4col_fullpair <<<grid_1677m, block_1677m >>> (dev_xor_result, dev_chrcode_mask, offset);
	}
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	validcnt = 0;

	// 8���Ԃ�̑g�������v�Z����
	for (int xor4idx = 0; xor4idx < (256 * 256 * 256); ++xor4idx) {

		// xor4idx�Ԗڂ̑g�����ɂ��Ď���4�����v�Z����
		calcxor8 << < 256 * 256, 256 >> > (dev_xor_result8, dev_xor_result, xor4idx);
		cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

		// 12�����̑g�������v�Z����
		for (int xor8idx = 0; xor8idx < (256 * 256 * 256); ++xor8idx) {
			// xor8idx�Ԗڂ̑g�����ɂ��Ď���4�����v�Z����
			calcxor8 << < 256 * 256, 256 >> > (dev_xor_result12, dev_xor_result8, xor8idx);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

			// 13���ڂ̓��o��14���ڂ̗L���ȕ����R�[�h������
			calcxor13_validchk << < 256 * 256 * 256, 64 >> > (dev_31F9_is07h, dev_xor_result12, dev_chrcode_mask);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }


			cudaStatus = hipMemcpy(cpu_31F9_valid, dev_31F9_is07h, 64 * 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
			for (int chkrs = 0; chkrs < (256 * 256 * 256 * 64); ++chkrs) {
				validcnt += cpu_31F9_valid[chkrs];
			}

			printf("%d items\n", validcnt);
			validcnt = 0;
		}
	}
//	kerntest <<< dim3(256*256), dim3(256) >>> ();

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(cpu_result, dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }


Error:
	hipFree(cpu_31F9_valid);
	hipFree(dev_31F9_is07h);
	hipFree(dev_xor_result12);
	hipFree(dev_xor_result8);
	hipFree(dev_xor_result);
	hipFree(dev_chrcode_mask);

	return(cudaStatus);

}
#if 0
// �f�o�C�X�Ƃ̂����
hipError_t chkPass(
	unsigned char *result_holder,		// OUTPUT �v�Z���ʂ̎���
	string	chrdic,						// ������̃e�[�u��
	int		passlength,					// �p�X���[�h����
	unsigned char *passcolmns			// �p�X���[�h�̑�������p�J�E���^[PASSWORD_LEN_MAX = 16]
)
{
	hipError_t cudaStatus;

	unsigned char *dev_result = 0;		// ���茋��
	unsigned char *dev_candidate = 0;	// �ƍ��Ώۂ̃L�[�ƒ������̔z��
	unsigned char *dev_reference = 0;	// ����Ώۂ̃`�F�b�N�f�B�W�b�g

	unsigned char *dev_passnum = 0;		// �p�X���[�h�����p�J�E���^
	unsigned char *dev_lut_code = 0;	// �����ϊ��e�[�u��

	int dlen = chrdic.size();

	unsigned char reference[16] = {
		0x65, 0x94, 0x0E, 0xAC, 0xE9, 0x07, 0x33, 0x25,	// ���T���ׂ��`�F�b�N�f�B�W�b�g�Q
		0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00
	};


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	// �ƍ����ʂ͎����̕����핪
	cudaStatus = hipMalloc((void**)&dev_result, chrdic.size() * chrdic.size() * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �p�X���[�h��␔�͕����핪
	cudaStatus = hipMalloc((void**)&dev_candidate, chrdic.size() * chrdic.size() * sizeof(unsigned char) * PASSWORD_LEN_MAX);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �����ϊ��e�[�u��
	cudaStatus = hipMalloc((void**)&dev_lut_code, 256);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �����e�[�u��
	cudaStatus = hipMalloc((void**)&dev_passnum, PASSWORD_LEN_MAX);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �ƍ��Ώۂ̃`�F�b�N�f�B�W�b�g�͈���
	cudaStatus = hipMalloc((void**)&dev_reference, 16);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// Copy input vectors from host memory to GPU buffers.
	// �����R�[�h�ϊ��e�[�u��
	cudaStatus = hipMemcpy(dev_lut_code, g_chrcode, 256, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	// �J�E���^���當���R�[�h����
	chrconvKernel << <chrdic.size(), chrdic.size() >> >	(dev_candidate, dev_passnum, dev_lut_code, passlength, chrdic.size());
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "chrconvKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	cudaStatus = hipMemcpy(dev_reference, reference, 16, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipDeviceSynchronize();

	// Launch a kernel on the GPU with one thread for each element.
	checkPassKernel << <chrdic.size(), chrdic.size() >> > (dev_result, dev_candidate, dev_reference);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result_holder, dev_result, chrdic.size() * chrdic.size(), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

Error:
	hipFree(dev_result);
	hipFree(dev_candidate);
	hipFree(dev_reference);
	hipFree(dev_passnum);
	hipFree(dev_lut_code);

	return(cudaStatus);
}

#endif