#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <fstream>
#include <string>
#include <forward_list>
#include <numeric>
#include <conio.h>

__device__ inline unsigned char bitrev(unsigned int v)
{
	unsigned int wk = __brev(v);
	return (wk >> 24) & 0xFF;
}

__device__ inline unsigned char adc(unsigned char &c, unsigned char vl, unsigned char vr)
{
	unsigned short wk = vl + vr + c;
	c = (wk & 0x0100) ? 1 : 0;
	return (unsigned char)(wk & 0xFF);
}

__device__ inline unsigned char ror(unsigned char &c, unsigned char v)
{
	unsigned char wc = c * 0x80;
	c = v & 0x01;
	return (unsigned char)((v >> 1) | wc);
}

__device__ inline unsigned char bitcnt(unsigned char v)
{
	return ((unsigned char)__popc(v));
}

__device__ unsigned char calcstep(const unsigned char chr, unsigned char *_31F4x)
{
	unsigned char	bchr, c;
	const unsigned char bmask_31f4[2] = { 0x00, 0x84 };
	const unsigned char bmask_31f5[2] = { 0x00, 0x08 };
	unsigned char wk31f4(_31F4x[0]), wk31f5(_31F4x[1]), wk31fa;

	bchr = bitrev(chr);
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	_31F4x[0] = wk31f4;							// 4
	_31F4x[1] = wk31f5;							// 5
//	_31F4x[2] = 0x0E;							// 6 ������

	c = (wk31f4 >= 0xE5) ? 1 : 0;
	_31F4x[3] = adc(c, chr, _31F4x[3]);			// 7

	_31F4x[4] = adc(c, _31F4x[4], _31F4x[1]);	// 8

	_31F4x[5] = chr ^ _31F4x[5];				// 9

	wk31fa = ror(c, _31F4x[6]);
	_31F4x[6] = adc(c, wk31fa, chr);			// A

	_31F4x[7] += c + bitcnt(chr);				// B

	return (c);
}

// <<< dim3(64 * 64, 64), dim3(64) >>>
__global__ void calc_1_4col(
	unsigned char		*work4col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	unsigned char		*valid,			// �L���ȕ����R�[�h�ō\�����ꂽ��
	const unsigned char *chrmask		// �L�������R�[�h���̔���p�r�b�g�}�X�N
)
{
	const unsigned int idx = ((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x;
	const unsigned int gblidx = idx * 8;

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;

	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	/*$31F4*/	work4col[gblidx + 0] = 0;
	/*$31F5*/	work4col[gblidx + 1] = 0;
	/*$31F6*/	work4col[gblidx + 2] = 0x0E;	// �Œ�l�A�s�g�p
	/*$31F7*/	work4col[gblidx + 3] = 0;
	/*$31F8*/	work4col[gblidx + 4] = 0;
	/*$31F9*/	work4col[gblidx + 5] = 0;
	/*$31FA*/	work4col[gblidx + 6] = 1;
	/*$31FB*/	work4col[gblidx + 7] = 0;

	calcstep(chr1, &work4col[gblidx]);
	calcstep(chr2, &work4col[gblidx]);
	calcstep(chr3, &work4col[gblidx]);
	calcstep(chr4, &work4col[gblidx]);

	valid[idx] = chrmask[chr1] | chrmask[chr2] | chrmask[chr3] | chrmask[chr4];
}

// ����܂ł̌��̒l���g���Ēǉ���4�����̌v�Z�l�����߂�
__global__ void calc_4col(
	unsigned char		*work8col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	unsigned char		*unvalid8,		// �����������܂܂�Ă�����true
	const unsigned char	*work4col,		// ����܂łɌv�Z���Ă������Z����
	const unsigned char *chrmask,		// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const unsigned int	idx4			// work4col�ɑΉ�����C���f�b�N�X�l
)
{
	// �X���b�h�ԍ����珑�����ݐ�C���f�b�N�X���v�Z
	const unsigned int idx = ((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x;
	const unsigned int gblidx = idx * 8;

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;
	const unsigned int	rdidx = idx4 * 8;

	/*$31F4*/	work8col[gblidx + 0] = work4col[rdidx + 0];
	/*$31F5*/	work8col[gblidx + 1] = work4col[rdidx + 1];
	/*$31F6*/	work8col[gblidx + 2] = work4col[rdidx + 2];
	/*$31F7*/	work8col[gblidx + 3] = work4col[rdidx + 3];
	/*$31F8*/	work8col[gblidx + 4] = work4col[rdidx + 4];
	/*$31F9*/	work8col[gblidx + 5] = work4col[rdidx + 5];
	/*$31FA*/	work8col[gblidx + 6] = work4col[rdidx + 6];
	/*$31FB*/	work8col[gblidx + 7] = work4col[rdidx + 7];

	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	calcstep(chr1, &work8col[gblidx]);
	calcstep(chr2, &work8col[gblidx]);
	calcstep(chr3, &work8col[gblidx]);
	calcstep(chr4, &work8col[gblidx]);

	unvalid8[idx] = chrmask[chr1] | chrmask[chr2] | chrmask[chr3] | chrmask[chr4];
}

// �c��̂P�����o�ƍŏI���܂ł̃`�F�b�N�f�B�W�b�g�v�Z�A�L������
__global__ void calclast_validate(
	unsigned char		*result,	// OUT  14�����̊e�f�[�^�v�Z����
	unsigned char		*valid,		// OUT  �p�X���[�h�Ƃ��Đ��藧�Ȃ�1(true)�łȂ����0(false)
	const unsigned char	*work,		// IN	12���ڂ܂łœ��o���ꂽ�r���l
	const unsigned int	workidx		// IN	blockIdx.x�̃I�t�Z�b�g
)
{
	const unsigned int	wkidx = workidx * 8;
	const unsigned int	gblidx = ((blockIdx.x * blockDim.x) + threadIdx.x) * 8;
	const unsigned char col13 = blockIdx.x;			// �u���b�Nidx.x��13���ڂ̕����R�[�h���
	const unsigned char col14 = threadIdx.x;		// �X���b�hidx.x��14���ڂ̕����R�[�h���

	/*$31F4*/	result[gblidx + 0] = work[wkidx + 0];
	/*$31F5*/	result[gblidx + 1] = work[wkidx + 1];
	/*$31F6*/	result[gblidx + 2] = work[wkidx + 2];
	/*$31F7*/	result[gblidx + 3] = work[wkidx + 3];
	/*$31F8*/	result[gblidx + 4] = work[wkidx + 4];
	/*$31F9*/	result[gblidx + 5] = work[wkidx + 5];
	/*$31FA*/	result[gblidx + 6] = work[wkidx + 6];
	/*$31FB*/	result[gblidx + 7] = work[wkidx + 7];

	const unsigned char chr13(col13 & 0x3F), chr14(col14 & 0x3F);

	calcstep(chr13, &result[gblidx]);
	calcstep(chr14, &result[gblidx]);

	bool	judge = true
		&& (result[gblidx + 0] == 0x65)	// $31F4
		&& (result[gblidx + 1] == 0x94)	// $31F5
		&& (result[gblidx + 2] == 0x0E)	// $31F6
		&& (result[gblidx + 3] == 0xAC)	// $31F7
		&& (result[gblidx + 4] == 0xE9)	// $31F8
		&& (result[gblidx + 5] == 0x07)	// $31F9
		&& (result[gblidx + 6] == 0x33)	// $31FA
		&& (result[gblidx + 7] == 0x25)	// $31FB
		;

	valid[(blockIdx.x * blockDim.x) + threadIdx.x] = judge;
}

using namespace std;

const unsigned char charvalidmask[64] = {
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,	// 0x06��0x07������
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x01,0x01,0x01,0x01,0x01,0x01,0x01,0x01,	// 0x38�ȏ������
};

const unsigned char chrcode[256] =
{
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xF0,0x2D,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0x2B,0xFF,0xFF,
	0x25,0x04,0x0C,0x14,0x1C,0x24,0x05,0x0D,0x15,0x1D,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xF0,0x00,0x08,0x10,0x18,0x20,0x28,0x30,0x01,0x09,0x11,0x19,0x21,0x29,0x31,0x02,
	0x0A,0x12,0x1A,0x22,0x2A,0x32,0x03,0x0B,0x13,0x1B,0x23,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0x33,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0x35,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0x2C,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0x34,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
	0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,0xFF,
};

bool countfrompass(string pass, unsigned int *st1_4, unsigned int *st5_8, unsigned int *st9_12, unsigned int *st13_14)
{
	// 14���ł��邱��
	if (pass.size() != 14) { return (false); }

	// �L�����������ō\������邱��
	for (auto ch : pass) { if (chrcode[ch] == 0xFF) { return (false); } }

	unsigned int stwk;
	stwk   = 0; stwk += chrcode[pass[ 0]];
	stwk <<= 6; stwk += chrcode[pass[ 1]];
	stwk <<= 6; stwk += chrcode[pass[ 2]];
	stwk <<= 6; stwk += chrcode[pass[ 3]];
	*st1_4 = stwk;
	stwk   = 0; stwk += chrcode[pass[ 4]];
	stwk <<= 6; stwk += chrcode[pass[ 5]];
	stwk <<= 6; stwk += chrcode[pass[ 6]];
	stwk <<= 6; stwk += chrcode[pass[ 7]];
	*st5_8 = stwk;
	stwk   = 0; stwk += chrcode[pass[ 8]];
	stwk <<= 6; stwk += chrcode[pass[ 9]];
	stwk <<= 6; stwk += chrcode[pass[10]];
	stwk <<= 6; stwk += chrcode[pass[11]];
	*st9_12 = stwk;
	stwk   = 0; stwk += chrcode[pass[12]];
	stwk <<= 6; stwk += chrcode[pass[13]];
	*st13_14 = stwk;

	return true;
}

hipError_t chkthread( forward_list<string>	&, unsigned int, unsigned int, unsigned int, unsigned int);

int main(int argc, char *argv[])
{
	hipError_t				cudaStatus;
	forward_list<string>	vPasswordList;
	unsigned int			sf1_4(0), sf5_8(0), sf9_12(0), sf13_14(0);
	string					filename, startpass;
	ofstream				writefile;

	switch (argc) {
	case 3: // �T���J�n������w�肠��
		startpass = argv[2];
		{
			bool valid = countfrompass(startpass, &sf1_4, &sf5_8, &sf9_12, &sf13_14);
			if (!valid) {
				fprintf(stderr, "�p�X���[�h�T���J�n�̎w�肪�Ԉ���Ă��܂��B\n");
				return (-1);
			}
		}
		// no break
	case 2:	// �o�̓t�@�C�����w�肠��
		filename = argv[1];
		break;
	case 1:
	default:
		fprintf(stderr, "unpass outfilename.txt [startpass]\n");
		return (-1);
	}

	writefile.open(filename, ios::out);
	if (!writefile.fail()) {
		printf(	"�t�@�C���A�N�Z�X���\�ł��邱�Ƃ��m�F�ł��܂���\n"
				"��U�N���[�Y���܂��B\n"
				"�w�肵���t�@�C���ɂ͐G�炸�ɂ��҂���������\n"
		);
	}
	writefile.close();

	chkthread(vPasswordList, sf1_4, sf5_8, sf9_12, sf13_14);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	printf("�\�[�g��...");
	vPasswordList.sort();
	printf("����\n");

	size_t listlen = distance(vPasswordList.begin(), vPasswordList.end());
	if (listlen > 0) {
		cout << listlen << "��������܂����B" << endl;

		printf("�t�@�C���o�͒�...\n");
		writefile.open(filename, ios::out);
		if (!writefile.fail()) {
			printf("�t�@�C�����J���܂����B\n"
				"�w�肵���t�@�C���ɂ͐G�炸�ɂ��҂����������B\n"
			);
			for (auto it : vPasswordList) {
				writefile << it << endl;
			}
		}
		writefile.close();
	}
	else {
		cout << "�w��͈͂ł͌�����܂���ł����B" << endl;
		cout << "�t�@�C���o�͂̓X�L�b�v���܂��B" << endl;

	}
	printf("����\n");

	return 0;
}

// �J�E���^����p�X������ւ̕ϊ�
void count2pass4(unsigned int count, char *pass)
{
	static const char _dict[] = { "AHOV16  BIPW27  CJQX38  DKRY49  ELSZ50  FMT-�!  GNU.Ѻ          " };
	pass[3] = _dict[count & 0x3F];	count >>= 6;
	pass[2] = _dict[count & 0x3F];	count >>= 6;
	pass[1] = _dict[count & 0x3F];	count >>= 6;
	pass[0] = _dict[count & 0x3F];
}
void count2pass2(unsigned int count, char *pass)
{
	static const char _dict[] = { "AHOV16  BIPW27  CJQX38  DKRY49  ELSZ50  FMT-�!  GNU.Ѻ          " };
	pass[1] = _dict[count & 0x3F];	count >>= 6;
	pass[0] = _dict[count & 0x3F];
}


hipError_t chkthread(
	forward_list<string>	&vPasswordList,
	unsigned int			stidx1,
	unsigned int			stidx2,
	unsigned int			stidx3,
	unsigned int			stidx4
	)
{
	hipError_t cudaStatus;
	unsigned char *dev_chrcode_mask = 0;	// �L�������R�[�h����p�}�X�N		�L����0x00�A������0x80�B
	unsigned char *dev_validpass = 0;		// �v�Z�������Ȃ��1������

	unsigned char *cpu_validpass = 0;		// �L���p�X���[�h���茋��(CPU��)
	unsigned char *cpu_calcresult = 0;		// �v�Z���ʊm�F�p

	unsigned char *cpu_unvalid4 = 0;		// �L�������ō\������Ă��邩���(�擪4��)
	unsigned char *cpu_unvalid8 = 0;		// �L�������ō\������Ă��邩���(5-8��)
	unsigned char *cpu_unvalid12 = 0;		// �L�������ō\������Ă��邩���(9-12��)

	unsigned char *dev_unvalid4 = 0;		// �L�������ō\������Ă��邩���(�擪4��)
	unsigned char *dev_unvalid8 = 0;		// �L�������ō\������Ă��邩���(5-8��)
	unsigned char *dev_unvalid12 = 0;		// �L�������ō\������Ă��邩���(9-12��)

	unsigned char *dev_res4 = 0;			// �擪4���Ԃ�̌v�Z���ʂƃL�����[���
	unsigned char *dev_res8 = 0;			// 5-8���܂ł̌v�Z���ʂƃL�����[���
	unsigned char *dev_res12 = 0;			// 9-12���܂Ōv�Z���ʂƃL�����[���
	unsigned char *dev_result = 0;			// 14���̌v�Z����

	unsigned long long validcnt = 0;		// �`�F�b�N�f�B�W�b�g��ʂ����p�X���[�h�̌�
	unsigned int  searchidx1;				// �T���J�n�ʒu(�擪4��)
	unsigned int  searchidx5;				// �T���J�n�ʒu(5-8��)
	unsigned int  searchidx9;				// �T���J�n�ʒu(9-12��)
	unsigned int  searchidx13;				// �T���J�n�ʒu(13-14��)

	char	passstr[15] = { "AAAAAAAAAAAAAA" };

	cudaStatus = hipSetDevice(0);		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }
	
	// ���o�p�e�[�u��(const)
	cudaStatus = hipMalloc((void**)&dev_chrcode_mask, 64 * sizeof(unsigned char));						if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 4�����̊e�f�[�^�Z�o�p�̈�m��
	cudaStatus = hipMalloc((void**)&dev_res4, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res8, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res12, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_result, 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_validpass, 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_unvalid4, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_unvalid8, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_unvalid12, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }


	cudaStatus = hipMemcpy(dev_chrcode_mask, charvalidmask, 64 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_validpass, 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_calcresult, 64 * 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_unvalid4, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_unvalid8, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_unvalid12, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �擪�S���S�g�݂��킹�̌v�Z
	calc_1_4col <<< dim3(64 * 64, 64), dim3(64) >>> (dev_res4, dev_unvalid4, dev_chrcode_mask);	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calu14col!\n", cudaStatus); goto Error; }
	cudaStatus = hipMemcpy(cpu_unvalid4, dev_unvalid4, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

	printf("\n�S�T���J�n");
	validcnt = 0;
	for (searchidx1 = stidx1; searchidx1 < (64 * 64 * 64 * 64); ++searchidx1) {
		// �����������܂܂�Ă����猟���Ώۏ��O
		if (cpu_unvalid4[searchidx1]) {
			continue;
		}
		count2pass4(searchidx1, &passstr[0]);
		printf("\nTotal %lld items\n", validcnt);
		printf("\n1-4 Loop %5.3f%% %.4s", (float)searchidx1 / (64 * 64 * 64 * 64), &passstr[0]);
		
		// 5-8���̑g�������v�Z����
		calc_4col << < dim3(64 * 64, 64), dim3(64) >> > (dev_res8, dev_unvalid8, dev_res4, dev_chrcode_mask, searchidx1);
		cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
		cudaStatus = hipMemcpy(cpu_unvalid8, dev_unvalid8, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

		for (searchidx5 = stidx2; searchidx5 < (64 * 64 * 64 * 64); ++searchidx5) {
			if (cpu_unvalid8[searchidx5]) {
				continue;
			}
			count2pass4(searchidx5, &passstr[4]);
			printf("\nTotal %lld items\n", validcnt);
			printf("\n5-8 Loop %5.3f%% %.4s", (float)searchidx5 / (64 * 64 * 64 * 64), &passstr[4]);
			// 9�`12�����̑g�������v�Z����
			calc_4col << < dim3(64 * 64, 64), 64 >> > (dev_res12, dev_unvalid12, dev_res8, dev_chrcode_mask, searchidx5);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
			cudaStatus = hipMemcpy(cpu_unvalid12, dev_unvalid12, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

			for (searchidx9 = stidx3; searchidx9 < (64 * 64 * 64 * 64); ++searchidx9) {
				if (cpu_unvalid12[searchidx9]) {
					continue;
				}
				count2pass4(searchidx9, &passstr[8]);
				// 13,14���ڂ̓��o�ƃ`�F�b�N�f�B�W�b�g���ʂ��Ă��邩�̔���
				calclast_validate << < 64, 64 >> > (dev_result, dev_validpass, dev_res12, searchidx9);
				cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching validation!\n", cudaStatus); goto Error; }

				cudaStatus = hipMemcpy(cpu_validpass, dev_validpass, 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
				for (searchidx13 = stidx4; searchidx13 < (64 * 64); ++searchidx13) {
					if (!cpu_validpass[searchidx13]) continue;
					cudaStatus = hipMemcpy(cpu_calcresult, dev_result, (64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
					count2pass2(searchidx13, &passstr[12]);

					vPasswordList.push_front(string(passstr));

					printf("\n%s | ", passstr);
					for (int vidx = 0; vidx < 8; ++vidx) {
						printf("%02X ", cpu_calcresult[searchidx13 * 8 + vidx]);
					}
					validcnt += cpu_validpass[searchidx13];
				}

				// ESC�L�[���̓`�F�b�N
				if (_kbhit() && (_getch() == 27)) {
					printf("\nChecked up to the item \"%s\"."
					"\n�T����ł��؂�܂���\n\n", passstr);
					goto FIN;
				}
			}
		}
	}
	printf("\n�T������\n\n");
FIN:
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(cpu_result, dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }


Error:
	hipFree(dev_chrcode_mask);
	hipFree(dev_validpass);

	hipFree(cpu_validpass);
	hipFree(cpu_calcresult);

	hipFree(dev_unvalid4);
	hipFree(dev_unvalid8);
	hipFree(dev_unvalid12);

	hipFree(dev_res4);
	hipFree(dev_res8);
	hipFree(dev_res12);
	hipFree(dev_result);

	hipFree(dev_unvalid4);
	hipFree(dev_unvalid8);
	hipFree(dev_unvalid12);

	return(cudaStatus);

}
