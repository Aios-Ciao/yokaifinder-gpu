#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <stdio.h>

#define PASSWORD_LEN		(14)
#define PASSWORD_LEN_MAX	(16)

// �p�X���[�h�̌v�Z�J�[�l��
__global__ void checkPassKernel(
	unsigned char *result,			// OUTPUT	�p�X������TRUE��Ԃ���	[x*y]
	const unsigned char*candidate,	// INPUT	�p�X���[�h���(�����R�[�h�ϊ��ς�) [x*y][16]
	const unsigned char*refer_chk	// INPUT	����p�`�F�b�N�f�B�W�b�g	[x*y][16]
)
{
	bool result_b(false), match_flg;
	int a(0), c(0), cb(0), chridx, strlength;
	unsigned char mem20(0), mem21(0);
	unsigned char mem50(0), mem51(0), mem52(0), mem53(1), mem54(0);

	int th_x = threadIdx.x;		// 
//	int th_y = threadIdx.y;

	auto ror = [&c](unsigned char v)
	{
		unsigned char wc = c;

		c = v & 0x01;
		return (unsigned char)((v >> 1) | (wc << 7));
	};
	auto adc = [&c](unsigned char lhs, unsigned char rhs)
	{
		unsigned short v;

		v = (unsigned short)lhs + (unsigned short)rhs + (unsigned short)c;
		c = (v > 0xFF) ? 1 : 0;

		return (v & 0xFF);
	};
	auto bitcnt = [](unsigned long int v)
	{
			v = (v & 0x55555555) + (v >> 1 & 0x55555555);
			v = (v & 0x33333333) + (v >> 2 & 0x33333333);
			v = (v & 0x0f0f0f0f) + (v >> 4 & 0x0f0f0f0f);
			v = (v & 0x00ff00ff) + (v >> 8 & 0x00ff00ff);
		return  (v & 0x0000ffff) + (v >> 16 & 0x0000ffff);
	};

	strlength = refer_chk[2];
	for (chridx = 0; chridx < strlength; ++chridx) {
		a = candidate[(th_x * PASSWORD_LEN_MAX) + chridx];

		// calc checkdigit1	(���[�v�W�J���Ă���)
//		for (int bit = 7; bit >= 0; --bit) {
//			c = (a >> bit) & 1;
//			mem20 = ror(mem20);
//			mem21 = ror(mem21);
//
//			if (c) {
//				mem20 ^= 0x84;
//				mem21 ^= 0x08;
//			}
//		}
		c = ((a & 0x80) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x40) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x20) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x10) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x08) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x04) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x02) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		c = ((a & 0x01) != 0);
		mem20 = ror(mem20);
		mem21 = ror(mem21);
		cb = c ? 0x84 : 0x00;
		mem20 ^= cb;
		cb = c ? 0x08 : 0x00;
		mem21 ^= cb;

		// calc checkdigit2
		c = (mem20 >= 0xE5) ? 1 : 0;
		mem50 = adc(a, mem50);
		mem51 = adc(mem51, mem21);
		// calc checkdigit3
		mem52 ^= a;
		// calc checkdigit4
		{
			unsigned char v = ror(mem53);
			mem53 = adc(v, a);
		}
		// calc checkdigit5
		mem54 += (unsigned char)(c + bitcnt(a));
	}
	
	result_b = true;
	match_flg = (mem20 == refer_chk[0]);
	result_b = result_b && match_flg;
	match_flg = (mem21 == refer_chk[1]);
	result_b = result_b && match_flg;
	match_flg = (mem50 == refer_chk[3]);
	result_b = result_b && match_flg;
	match_flg = (mem51 == refer_chk[4]);
	result_b = result_b && match_flg;
	match_flg = (mem52 == refer_chk[5]);
	result_b = result_b && match_flg;
	match_flg = (mem53 == refer_chk[6]);
	result_b = result_b && match_flg;
	match_flg = (mem54 == refer_chk[7]);
	result_b = result_b && match_flg;

	result[th_x] = result_b;
}


// 2�����̃p�X���[�h���𐶐�����
__global__ void chrconvKernel(
	unsigned char *candidate,			// OUTPUT	����������f�[�^
	const unsigned char *basecode,		// INPUT	�Œ蕔���Ƃ��ĎQ�Ƃ��錅���[16]
	const unsigned char *tbl_chrdict,	// INPUT	�����R�[�h�e�[�u��[256]
	const unsigned int	passlen,		// INPUT	�ϊ��Ώۂ̌���(�S��)
	const unsigned int	dictlen			// INPUT	�����퐔
)
{
	int th_x = threadIdx.x;

	for (int i = 0; i < (passlen - 1); ++i) {
		candidate[(PASSWORD_LEN_MAX * th_x) + i] = tbl_chrdict[basecode[i]];
	}
}

__global__ void make_4col_fullpair(
	unsigned char *pass4col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*4����͂��܂�4byte
	unsigned char *chrmask,			// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const int  offset				// blockIdx��z�ɗ������鉺��
)
{
	unsigned int	blocksize = blockDim.x * blockDim.y * blockDim.z;
	unsigned int	thidx = (threadIdx.z * (blockDim.x * blockDim.y)) + (threadIdx.y * blockDim.x) + threadIdx.x;
	unsigned int	gridx = ((gridDim.x * gridDim.y) * ((blockIdx.z + (offset*gridDim.z)))) + (gridDim.x * blockIdx.y) + blockIdx.x;
	unsigned int	gblidx = gridx * blocksize + thidx;

	unsigned char	col1 = blockIdx.z + (offset * gridDim.z);
	unsigned char	col2 = blockIdx.y;
	unsigned char	col3 = blockIdx.x * (blockDim.z * blockDim.y) + threadIdx.z * blockDim.y + threadIdx.y;
	unsigned char	col4 = threadIdx.x;
	
	unsigned char unvalid;

	unvalid = chrmask[col1] | chrmask[col2] | chrmask[col3] | chrmask[col4];

	pass4col[gblidx] = unvalid | (col1 ^ col2 ^ col3 ^ col4);
}

// ����܂ł�XOR���ʂ�4���Ԃ��XOR�����߂�
__global__ void calcxor8(
	unsigned char		*xor8,		// ���Z����
	const unsigned char *xor4,		// ����܂ł̉��Z����
	const unsigned int	idxG		// xor8�̃C���f�b�N�X
)
{
	unsigned int idxH = (blockIdx.x * 256) + threadIdx.x;

	unsigned char W1 = xor4[idxG];
	unsigned char W2 = xor4[idxH];
	unsigned char enf = (W1 | W2) & 0x80;

	xor8[idxH] = enf | (W1 ^ W2);
}

// 13���ڂ̓��o��xor 0x07���ʂ̑Ó������f
__global__ void calcxor13_validchk(
	unsigned char		*valid14,		// 14���ڂ��L���Ȃ�1���i�[
	const unsigned char *xor12,			// 12���ڂ܂œ��o���ꂽXOR
	const unsigned char *chrmask		// �L���ȕ����R�[�h���ǂ���
)
{
	unsigned char col13 = xor12[blockIdx.x] ^ threadIdx.x;	// 13���ڂ̓��o 1677���v�f�̓��Y���l

	col13 |= chrmask[threadIdx.x];
	col13 ^= 0x07;

	valid14[(blockIdx.x*blockDim.x) + threadIdx.x] = !(col13 & 0x80);	// �����ȕ����R�[�h�������������
}

// �J�[�l���Ăяo���e�X�g�p
__global__ void kerntest()
{
}


using namespace std;

//hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
hipError_t chkPass(unsigned char *, string, int, unsigned char *);

const unsigned char g_chrcode[256] =
{
	0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0x2D,0,0,0,0,0,0,0,0,0,0,0,0x2B,0,0,
0x25,0x04,0x0C,0x14,0x1C,0x24,0x05,0x0D,0x15,0x1D,0,0,0,0,0,0,
0,0x00,0x08,0x10,0x18,0x20,0x28,0x30,0x01,0x09,0x11,0x19,0x21,0x29,0x31,0x02,
0x0A,0x12,0x1A,0x22,0x2A,0x32,0x03,0x0B,0x13,0x1B,0x23,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0x33,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0x35,0,0,0,0,0,
0,0,0,0,0,0x2C,0,0,0,0,0,0,0,0,0,0,
0,0x34,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,0,
};

//const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ" };
const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!.-nmc�Ѻ" };

const unsigned char charvalidmask[64] = { 
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,	// 0x06��0x07������
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
	0x00,0x00,0x00,0x00,0x00,0x00,0x80,0x80,
};

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
);

const unsigned long	WORKSIZE = 64 * 64 * 64 * 64;
static unsigned char result[WORKSIZE];

int main()
{
	chkthread(result, WORKSIZE);

#if 0
	unsigned char	pass_columns[PASSWORD_LEN_MAX] = { 0 };
	int passed = -1;

	hipError_t cudaStatus;

	const size_t	dictlen = sizeof(_dict);
	std::string chrtable(_dict, dictlen);
	unsigned char *chkresult = new unsigned char[dictlen * dictlen];

	const int		COLUMNS = 14;
	const int		refresh_col_n = 4;
	const int		chkcol = 8;
	unsigned char	chkkey;
	unsigned char	refresh_col;		// �i���\���p�̔��茅

	// �I������p���l��ޔ�
	chkkey = pass_columns[chkcol];
	refresh_col = pass_columns[refresh_col_n];

	while (pass_columns[chkcol] == chkkey) {

		if (refresh_col != pass_columns[refresh_col_n]) {

			for (int idx = 0; idx < PASSWORD_LEN; ++idx) {
				fprintf(stderr, "%c", _dict[pass_columns[idx]]);
			}
			fprintf(stderr, "\n");
			refresh_col = pass_columns[refresh_col_n];
		}

		hipError_t cudaStatus = chkPass(chkresult, chrtable, 14, pass_columns);
		if (cudaStatus != hipSuccess) { fprintf(stderr, "passcheck failed!"); return 1; }

		for (int idx = 0; idx < (chrtable.size() * chrtable.size()); ++idx) {
			if (chkresult[idx] != 0) {
				fprintf(stderr, "Passed %d\n", idx);
			}
		}

		// �I�������ɍ��v�����Ԃ𔻒肵����T���I��
		bool carry = true;
		for (int col = (2); col < (COLUMNS - 1); ++col) {
			//	for (int col = 2; col < (COLUMNS - 1); ++col) {
			pass_columns[col] += carry ? 1 : 0;
			carry = (pass_columns[col] == dictlen);
			pass_columns[col] = carry ? 0 : pass_columns[col];
			//	candidate[col] = dict[counters[col]];
		}
	}

	
	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	delete[] chkresult;
#endif
	return 0;
}

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
)
{
	hipError_t cudaStatus;
//	unsigned char *dev_result = 0;		// ���茋��
	unsigned char *dev_xor_result = 0;		// 4���Ԃ��xor����
	unsigned char *dev_xor_result8 = 0;		// 8���Ԃ��xor����(�e���|����)
	unsigned char *dev_xor_result12 = 0;	// 12���Ԃ��xor����(�e���|����)
	unsigned char *dev_31F9_is07h = 0;		// 13���ڂ܂�xor�����߂����ʂ�0x07��xor�����l(14���ڂ̕����R�[�h)���L���ȕ����R�[�h��

	unsigned char *dev_chrcode_mask = 0;	// �L�������R�[�h����p�}�X�N		�L����0x00�A������0x80�B

	unsigned char *cpu_31F9_valid = 0;
	unsigned long validcnt = 0;


//	dim3	 block_530m(48, 8, 2), grid_530m(3, 48, 6);	// ����48�� 4�����̕���v�Z�O���b�h�p	 5.3M items
	dim3	 block_1677m(64, 8, 2), grid_1677m(4, 64, 4);	// ����64�� 4�����̕���v�Z�O���b�h�p	16.8M items

	cudaStatus = hipSetDevice(0);																		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	// 4�������g������XOR�f�[�^����
	cudaStatus = hipMalloc((void**)&dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_xor_result8, 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_xor_result12, 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_31F9_is07h, 64 * 64 * 64 * 64 * 64 * sizeof(unsigned char));		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 64�����핪�̃R�[�h�L���������
	cudaStatus = hipMalloc((void**)&dev_chrcode_mask, 64 * sizeof(unsigned char));						if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMemcpy(dev_chrcode_mask, charvalidmask, 64 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_31F9_valid, 64 * 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 4�������g������XOR�f�[�^�v�Z
	for (int offset = 0; offset < 16; ++offset) {
		make_4col_fullpair <<<grid_1677m, block_1677m >>> (dev_xor_result, dev_chrcode_mask, offset);
	}
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	validcnt = 0;

	// 8���Ԃ�̑g�������v�Z����
	for (int xor4idx = 0; xor4idx < (256 * 256 * 256); ++xor4idx) {

		// xor4idx�Ԗڂ̑g�����ɂ��Ď���4�����v�Z����
		calcxor8 << < 256 * 256, 256 >> > (dev_xor_result8, dev_xor_result, xor4idx);
		cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

		// 12�����̑g�������v�Z����
		for (int xor8idx = 0; xor8idx < (256 * 256 * 256); ++xor8idx) {
			// xor8idx�Ԗڂ̑g�����ɂ��Ď���4�����v�Z����
			calcxor8 << < 256 * 256, 256 >> > (dev_xor_result12, dev_xor_result8, xor8idx);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

			// 13���ڂ̓��o��14���ڂ̗L���ȕ����R�[�h������
			calcxor13_validchk << < 256 * 256 * 256, 64 >> > (dev_31F9_is07h, dev_xor_result12, dev_chrcode_mask);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }


			cudaStatus = hipMemcpy(cpu_31F9_valid, dev_31F9_is07h, 64 * 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
			for (int chkrs = 0; chkrs < (256 * 256 * 256 * 64); ++chkrs) {
				validcnt += cpu_31F9_valid[chkrs];
			}

			printf("%d items\n", validcnt);
			validcnt = 0;
		}
	}
//	kerntest <<< dim3(256*256), dim3(256) >>> ();

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(cpu_result, dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }


Error:
	hipFree(cpu_31F9_valid);
	hipFree(dev_31F9_is07h);
	hipFree(dev_xor_result12);
	hipFree(dev_xor_result8);
	hipFree(dev_xor_result);
	hipFree(dev_chrcode_mask);

	return(cudaStatus);

}
#if 0
// �f�o�C�X�Ƃ̂����
hipError_t chkPass(
	unsigned char *result_holder,		// OUTPUT �v�Z���ʂ̎���
	string	chrdic,						// ������̃e�[�u��
	int		passlength,					// �p�X���[�h����
	unsigned char *passcolmns			// �p�X���[�h�̑�������p�J�E���^[PASSWORD_LEN_MAX = 16]
)
{
	hipError_t cudaStatus;

	unsigned char *dev_result = 0;		// ���茋��
	unsigned char *dev_candidate = 0;	// �ƍ��Ώۂ̃L�[�ƒ������̔z��
	unsigned char *dev_reference = 0;	// ����Ώۂ̃`�F�b�N�f�B�W�b�g

	unsigned char *dev_passnum = 0;		// �p�X���[�h�����p�J�E���^
	unsigned char *dev_lut_code = 0;	// �����ϊ��e�[�u��

	int dlen = chrdic.size();

	unsigned char reference[16] = {
		0x65, 0x94, 0x0E, 0xAC, 0xE9, 0x07, 0x33, 0x25,	// ���T���ׂ��`�F�b�N�f�B�W�b�g�Q
		0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00, 0x00
	};


	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }

	// �ƍ����ʂ͎����̕����핪
	cudaStatus = hipMalloc((void**)&dev_result, chrdic.size() * chrdic.size() * sizeof(unsigned char));
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �p�X���[�h��␔�͕����핪
	cudaStatus = hipMalloc((void**)&dev_candidate, chrdic.size() * chrdic.size() * sizeof(unsigned char) * PASSWORD_LEN_MAX);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �����ϊ��e�[�u��
	cudaStatus = hipMalloc((void**)&dev_lut_code, 256);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �����e�[�u��
	cudaStatus = hipMalloc((void**)&dev_passnum, PASSWORD_LEN_MAX);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �ƍ��Ώۂ̃`�F�b�N�f�B�W�b�g�͈���
	cudaStatus = hipMalloc((void**)&dev_reference, 16);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// Copy input vectors from host memory to GPU buffers.
	// �����R�[�h�ϊ��e�[�u��
	cudaStatus = hipMemcpy(dev_lut_code, g_chrcode, 256, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	// �J�E���^���當���R�[�h����
	chrconvKernel << <chrdic.size(), chrdic.size() >> >	(dev_candidate, dev_passnum, dev_lut_code, passlength, chrdic.size());
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "chrconvKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	cudaStatus = hipMemcpy(dev_reference, reference, 16, hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipDeviceSynchronize();

	// Launch a kernel on the GPU with one thread for each element.
	checkPassKernel << <chrdic.size(), chrdic.size() >> > (dev_result, dev_candidate, dev_reference);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(result_holder, dev_result, chrdic.size() * chrdic.size(), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

Error:
	hipFree(dev_result);
	hipFree(dev_candidate);
	hipFree(dev_reference);
	hipFree(dev_passnum);
	hipFree(dev_lut_code);

	return(cudaStatus);
}

#endif