#include "hip/hip_runtime.h"
#include <iostream>
#include "hip/hip_runtime.h"
#include ""
#include <string>
#include <stdio.h>

__device__ inline unsigned char bitrev(unsigned int v)
{
	unsigned int wk = __brev(v);
	return (wk >> 24) & 0xFF;
}

__device__ inline unsigned char adc(unsigned char &c, unsigned char vl, unsigned char vr)
{
	unsigned short wk = vl + vr + c;
	c = (wk & 0x0100) ? 1 : 0;
	return (unsigned char)(wk & 0xFF);
}

__device__ inline unsigned char ror(unsigned char &c, unsigned char v)
{
	unsigned char wc = c * 0x80;
	c = v & 0x01;
	return (unsigned char)((v >> 1) | wc);
}

__device__ inline unsigned char bitcnt(unsigned char v)
{
	return ((unsigned char)__popc(v));
}

__device__ unsigned char calcstep(const unsigned char chr, unsigned char *_31F4x)
{
	unsigned char	bchr, c;
	const unsigned char bmask_31f4[2] = { 0x00, 0x84 };
	const unsigned char bmask_31f5[2] = { 0x00, 0x08 };
	unsigned char wk31f4(_31F4x[0]), wk31f5(_31F4x[1]), wk31fa;

	bchr = bitrev(chr);
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	c = bchr & 0x01; bchr >>= 1; wk31f4 = ror(c, wk31f4); wk31f5 = ror(c, wk31f5); wk31f4 ^= bmask_31f4[c]; wk31f5 ^= bmask_31f5[c];
	_31F4x[0] = wk31f4;							// 4
	_31F4x[1] = wk31f5;							// 5
//	_31F4x[2] = 0x0E;							// 6 ������

	c = (wk31f4 >= 0xE5) ? 1 : 0;
	_31F4x[3] = adc(c, chr, _31F4x[3]);			// 7

	_31F4x[4] = adc(c, _31F4x[4], _31F4x[1]);	// 8

	_31F4x[5] = chr ^ _31F4x[5];				// 9

	wk31fa = ror(c, _31F4x[6]);
	_31F4x[6] = adc(c, wk31fa, chr);			// A

	_31F4x[7] += c + bitcnt(chr);				// B

	return (c);
}

// <<< dim3(64 * 64, 64), dim3(64) >>>
__global__ void calc_1_4col(
	unsigned char		*work4col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	unsigned char		*valid,			// �L���ȕ����R�[�h�ō\�����ꂽ��
	const unsigned char *chrmask		// �L�������R�[�h���̔���p�r�b�g�}�X�N
)
{
	const unsigned int idx = ((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x;
	const unsigned int gblidx = idx * 8;

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;

	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	/*$31F4*/	work4col[gblidx + 0] = 0;
	/*$31F5*/	work4col[gblidx + 1] = 0;
	/*$31F6*/	work4col[gblidx + 2] = 0x0E;	// �Œ�l�A�s�g�p
	/*$31F7*/	work4col[gblidx + 3] = 0;
	/*$31F8*/	work4col[gblidx + 4] = 0;
	/*$31F9*/	work4col[gblidx + 5] = 0;
	/*$31FA*/	work4col[gblidx + 6] = 1;
	/*$31FB*/	work4col[gblidx + 7] = 0;

	calcstep(chr1, &work4col[gblidx]);
	calcstep(chr2, &work4col[gblidx]);
	calcstep(chr3, &work4col[gblidx]);
	calcstep(chr4, &work4col[gblidx]);

	valid[idx] = chrmask[chr1] | chrmask[chr2] | chrmask[chr3] | chrmask[chr4];
}

// ����܂ł̌��̒l���g���Ēǉ���4�����̌v�Z�l�����߂�
__global__ void calc_4col(
	unsigned char		*work8col,		// �i�[�� �����̗v�f�͋��߂��O���[�o���C���f�b�N�X*8����͂��܂�8byte
	unsigned char		*unvalid8,		// �����������܂܂�Ă�����true
	const unsigned char	*work4col,		// ����܂łɌv�Z���Ă������Z����
	const unsigned char *chrmask,		// �L�������R�[�h���̔���p�r�b�g�}�X�N
	const unsigned int	idx4			// work4col�ɑΉ�����C���f�b�N�X�l
)
{
	// �X���b�h�ԍ����珑�����ݐ�C���f�b�N�X���v�Z
	const unsigned int idx = ((blockIdx.y * gridDim.x) + blockIdx.x) * blockDim.x + threadIdx.x;
	const unsigned int gblidx = idx * 8;

	const unsigned char	col1 = blockIdx.y;
	const unsigned char	col2 = blockIdx.x >> 6;
	const unsigned char	col3 = blockIdx.x & 0x3F;
	const unsigned char	col4 = threadIdx.x;
	const unsigned int	rdidx = idx4 * 8;

	/*$31F4*/	work8col[gblidx + 0] = work4col[rdidx + 0];
	/*$31F5*/	work8col[gblidx + 1] = work4col[rdidx + 1];
	/*$31F6*/	work8col[gblidx + 2] = work4col[rdidx + 2];
	/*$31F7*/	work8col[gblidx + 3] = work4col[rdidx + 3];
	/*$31F8*/	work8col[gblidx + 4] = work4col[rdidx + 4];
	/*$31F9*/	work8col[gblidx + 5] = work4col[rdidx + 5];
	/*$31FA*/	work8col[gblidx + 6] = work4col[rdidx + 6];
	/*$31FB*/	work8col[gblidx + 7] = work4col[rdidx + 7];

	const unsigned char chr1(col1 & 0x3F), chr2(col2 & 0x3F), chr3(col3 & 0x3F), chr4(col4 & 0x3F);

	calcstep(chr1, &work8col[gblidx]);
	calcstep(chr2, &work8col[gblidx]);
	calcstep(chr3, &work8col[gblidx]);
	calcstep(chr4, &work8col[gblidx]);

	unvalid8[idx] = chrmask[chr1] | chrmask[chr2] | chrmask[chr3] | chrmask[chr4];
}

// �c��̂P�����o�ƍŏI���܂ł̃`�F�b�N�f�B�W�b�g�v�Z�A�L������
__global__ void calclast_validate(
	unsigned char		*result,	// OUT  14�����̊e�f�[�^�v�Z����
	unsigned char		*valid,		// OUT  �p�X���[�h�Ƃ��Đ��藧�Ȃ�1(true)�łȂ����0(false)
	const unsigned char	*work,		// IN	12���ڂ܂łœ��o���ꂽ�r���l
	const unsigned int	workidx		// IN	blockIdx.x�̃I�t�Z�b�g
)
{
	const unsigned int	wkidx = workidx * 8;
	const unsigned int	gblidx = ((blockIdx.x * blockDim.x) + threadIdx.x) * 8;
	const unsigned char col13 = blockIdx.x;			// �u���b�Nidx.x��13���ڂ̕����R�[�h���
	const unsigned char col14 = threadIdx.x;		// �X���b�hidx.x��14���ڂ̕����R�[�h���

	/*$31F4*/	result[gblidx + 0] = work[wkidx + 0];
	/*$31F5*/	result[gblidx + 1] = work[wkidx + 1];
	/*$31F6*/	result[gblidx + 2] = work[wkidx + 2];
	/*$31F7*/	result[gblidx + 3] = work[wkidx + 3];
	/*$31F8*/	result[gblidx + 4] = work[wkidx + 4];
	/*$31F9*/	result[gblidx + 5] = work[wkidx + 5];
	/*$31FA*/	result[gblidx + 6] = work[wkidx + 6];
	/*$31FB*/	result[gblidx + 7] = work[wkidx + 7];

	const unsigned char chr13(col13 & 0x3F), chr14(col14 & 0x3F);

	calcstep(chr13, &result[gblidx]);
	calcstep(chr14, &result[gblidx]);

	bool	judge = true
		&& (result[gblidx + 0] == 0x65)	// $31F4
		&& (result[gblidx + 1] == 0x94)	// $31F5
		&& (result[gblidx + 2] == 0x0E)	// $31F6
		&& (result[gblidx + 3] == 0xAC)	// $31F7
		&& (result[gblidx + 4] == 0xE9)	// $31F8
		&& (result[gblidx + 5] == 0x07)	// $31F9
		&& (result[gblidx + 6] == 0x33)	// $31FA
		&& (result[gblidx + 7] == 0x25)	// $31FB
		;

	valid[(blockIdx.x * blockDim.x) + threadIdx.x] = judge;
}

using namespace std;

// const char _dict[] = { "ABCDEFGHIJKLMNOPQRSTUVWXYZ0123456789!.-nmc�Ѻ" };
const unsigned char charvalidmask[64] = {
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,	// 0x06��0x07������
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
	0x00,0x00,0x00,0x00,0x00,0x00,0x01,0x01,
};

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
);

const unsigned long	WORKSIZE = 64 * 64 * 64 * 64;
static unsigned char result[WORKSIZE];

int main()
{
	hipError_t cudaStatus;

	chkthread(result, WORKSIZE);

	// hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceReset failed!"); return 1; }

	return 0;
}

hipError_t chkthread(
	unsigned char *cpu_result,
	unsigned long	item_length
)
{
	hipError_t cudaStatus;
	unsigned char *dev_chrcode_mask = 0;	// �L�������R�[�h����p�}�X�N		�L����0x00�A������0x80�B
	unsigned char *dev_validpass = 0;		// �v�Z�������Ȃ��1������

	unsigned char *cpu_validpass = 0;		// �L���p�X���[�h���茋��(CPU��)
	unsigned char *cpu_calcresult = 0;		// �v�Z���ʊm�F�p

	unsigned char *cpu_unvalid4 = 0;		// �L�������ō\������Ă��邩���(�擪4��)
	unsigned char *cpu_unvalid8 = 0;		// �L�������ō\������Ă��邩���(5-8��)
	unsigned char *cpu_unvalid12 = 0;		// �L�������ō\������Ă��邩���(9-12��)

	unsigned char *dev_unvalid4 = 0;		// �L�������ō\������Ă��邩���(�擪4��)
	unsigned char *dev_unvalid8 = 0;		// �L�������ō\������Ă��邩���(5-8��)
	unsigned char *dev_unvalid12 = 0;		// �L�������ō\������Ă��邩���(9-12��)

	unsigned char *dev_res4 = 0;			// �擪4���Ԃ�̌v�Z���ʂƃL�����[���
	unsigned char *dev_res8 = 0;			// 5-8���܂ł̌v�Z���ʂƃL�����[���
	unsigned char *dev_res12 = 0;			// 9-12���܂Ōv�Z���ʂƃL�����[���
	unsigned char *dev_result = 0;			// 14���̌v�Z����

	unsigned long long validcnt = 0;		// �`�F�b�N�f�B�W�b�g��ʂ����p�X���[�h�̌�

	static const char cvalid[2] = { '.', 'O' };	// ����\���p


	cudaStatus = hipSetDevice(0);		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?"); goto Error; }
	
	// ���o�p�e�[�u��(const)
	cudaStatus = hipMalloc((void**)&dev_chrcode_mask, 64 * sizeof(unsigned char));						if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// 4�����̊e�f�[�^�Z�o�p�̈�m��
	cudaStatus = hipMalloc((void**)&dev_res4, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res8, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_res12, 64 * 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_result, 64 * 64 * 64 * 8);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_validpass, 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	cudaStatus = hipMalloc((void**)&dev_unvalid4, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_unvalid8, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipMalloc((void**)&dev_unvalid12, 64 * 64 * 64 * 64);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }


	cudaStatus = hipMemcpy(dev_chrcode_mask, charvalidmask, 64 * sizeof(unsigned char), hipMemcpyHostToDevice);	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_validpass, 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_calcresult, 64 * 64 * 64 * 64 * 8 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	cudaStatus = hipHostMalloc((void**)&cpu_unvalid4, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_unvalid8, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }
	cudaStatus = hipHostMalloc((void**)&cpu_unvalid12, 64 * 64 * 64 * 64 * sizeof(unsigned char));	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMalloc failed!"); goto Error; }

	// �擪�S���S�g�݂��킹�̌v�Z
	calc_1_4col <<< dim3(64 * 64, 64), dim3(64) >>> (dev_res4, dev_unvalid4, dev_chrcode_mask);	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching calu14col!\n", cudaStatus); goto Error; }
	cudaStatus = hipMemcpy(cpu_unvalid4, dev_unvalid4, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

	//cudaStatus = hipMemcpy(cpu_calcresult, dev_res4, (64 * 64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
	//for (int chkrs = 0; chkrs < 16/*(64 * 64 * 64)*/; ++chkrs) {
	//	printf("\n");

	//	for (int vidx = 0; vidx < 8; ++vidx) {
	//		printf("%02x ", cpu_calcresult[chkrs * 8 + vidx]);
	//	}
	//}

	validcnt = 0;
	for (int xor4idx = 0; xor4idx < (64 * 64 * 64 * 64); ++xor4idx) {
		// �����������܂܂�Ă����猟���Ώۏ��O
		if (cpu_unvalid4[xor4idx]) {
			continue;
		}
		
		// 5-8���̑g�������v�Z����
		calc_4col << < dim3(64 * 64, 64), dim3(64) >> > (dev_res8, dev_unvalid8, dev_res4, dev_chrcode_mask, xor4idx);
		cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
		cudaStatus = hipMemcpy(cpu_unvalid8, dev_unvalid8, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

		//cudaStatus = hipMemcpy(cpu_calcresult, dev_res8, (64 * 64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
		//for (int chkrs = 0; chkrs < (64 * 64 * 64); ++chkrs) {
		//	printf("\n");

		//	for (int vidx = 0; vidx < 8; ++vidx) {
		//		printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
		//	}
		//}

		for (int xor8idx = 0; xor8idx < (64 * 64 * 64 * 64); ++xor8idx) {
			if (cpu_unvalid8[xor8idx]) {
				continue;
			}
			// 9�`12�����̑g�������v�Z����
			calc_4col << < dim3(64 * 64, 64), 64 >> > (dev_res12, dev_unvalid12, dev_res8, dev_chrcode_mask, xor8idx);
			cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }
			cudaStatus = hipMemcpy(cpu_unvalid12, dev_unvalid12, (64 * 64 * 64 * 64) * sizeof(unsigned char), hipMemcpyDeviceToHost);

			//cudaStatus = hipMemcpy(cpu_calcresult, dev_res12, (64 * 64 * 64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);
			//for (int chkrs = 0; chkrs < (64 * 64 * 64); ++chkrs) {
			//	printf("\n");

			//	for (int vidx = 0; vidx < 8; ++vidx) {
			//		printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
			//	}
			//}

			for (int xor12idx = 0; xor12idx < (64 * 64 * 64 * 64); ++xor12idx) {
				if (cpu_unvalid12[xor12idx]) {
					continue;
				}
				// 13,14���ڂ̓��o�ƃ`�F�b�N�f�B�W�b�g���ʂ��Ă��邩�̔���
				calclast_validate << < 64, 64 >> > (dev_result, dev_validpass, dev_res12, xor12idx);
				cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching validation!\n", cudaStatus); goto Error; }

				cudaStatus = hipMemcpy(cpu_validpass, dev_validpass, 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
				for (int chkrs = 0; chkrs < (64 * 64); ++chkrs) {
					if (!cpu_validpass[chkrs]) continue;
					cudaStatus = hipMemcpy(cpu_calcresult, dev_result, (64 * 64 * 8) * sizeof(unsigned char), hipMemcpyDeviceToHost);

					printf("\n");

					printf("%08X %08X %08X %04X | ", xor4idx, xor8idx, xor12idx, chkrs);
					for (int vidx = 0; vidx < 8; ++vidx) {
						printf("%02X ", cpu_calcresult[chkrs * 8 + vidx]);
					}
					printf("| %c ", cvalid[cpu_validpass[chkrs]]);
					validcnt += cpu_validpass[chkrs];
				}
		//		goto FIN;

			}
			printf("\n%lld items\n", validcnt);
		}
	}
FIN:
	printf("found %lld items\n", validcnt);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();			if (cudaStatus != hipSuccess) { fprintf(stderr, "checkPassKernel launch failed: %s\n", hipGetErrorString(cudaStatus)); goto Error; }

	// hipDeviceSynchronize waits for the kernel to finish, and returns
// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();		if (cudaStatus != hipSuccess) { fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus); goto Error; }

	// Copy output vector from GPU buffer to host memory.
//	cudaStatus = hipMemcpy(cpu_result, dev_xor_result, 64 * 64 * 64 * 64 * sizeof(unsigned char), hipMemcpyDeviceToHost);
//	if (cudaStatus != hipSuccess) { fprintf(stderr, "hipMemcpy failed!"); goto Error; }


Error:
	hipFree(dev_chrcode_mask);
	hipFree(dev_validpass);

	hipFree(cpu_validpass);
	hipFree(cpu_calcresult);

	hipFree(dev_unvalid4);
	hipFree(dev_unvalid8);
	hipFree(dev_unvalid12);

	hipFree(dev_res4);
	hipFree(dev_res8);
	hipFree(dev_res12);
	hipFree(dev_result);

	hipFree(dev_unvalid4);
	hipFree(dev_unvalid8);
	hipFree(dev_unvalid12);

	return(cudaStatus);

}
